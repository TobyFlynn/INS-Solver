#include "hip/hip_runtime.h"
#include "op_lib_cpp.h"
#include "op_cuda_rt_support.h"
#include "op_cuda_reduction.h"

#include "dg_compiler_defs.h"

template<int p, int dg_np, int dg_npf>
__device__ void _pmf_3d_mult_cells_part1_gpu(const int ind, const double *mMat, const double *eMat,
                      const double *J, const double *lx, const double *ly, const double *lz,
                      const double *out_tmp, const double *ux, const double *uy, const double *uz,
                      double *outx, double *outy, double *outz, double *out) {
  if(!(ind < dg_np))
    return;
  const double *mmat_mat = &mMat[(p - 1) * DG_NP * DG_NP];
  const double *emat_mat = &eMat[(p - 1) * DG_NUM_FACES * DG_NPF * DG_NP];

  double outx_t = 0.0;
  double outy_t = 0.0;
  double outz_t = 0.0;
  for(int j = 0; j < dg_np; j++) {
    int mat_ind = DG_MAT_IND(ind, j, dg_np, dg_np);
    outx_t += mmat_mat[mat_ind] * ux[j];
    outy_t += mmat_mat[mat_ind] * uy[j];
    outz_t += mmat_mat[mat_ind] * uz[j];
  }
  outx_t *= *J;
  outy_t *= *J;
  outz_t *= *J;
  double out_t = 0.0;
  for(int j = 0; j < DG_NUM_FACES * dg_npf; j++) {
    int mat_ind = DG_MAT_IND(ind, j, dg_np, DG_NUM_FACES * dg_npf);
    outx_t += emat_mat[mat_ind] * lx[j];
    outy_t += emat_mat[mat_ind] * ly[j];
    outz_t += emat_mat[mat_ind] * lz[j];
    out_t  += emat_mat[mat_ind] * out_tmp[j];
  }
  outx[ind] = outx_t;
  outy[ind] = outy_t;
  outz[ind] = outz_t;
  out[ind] = out_t;
}

template<int p, int dg_np>
__device__ void _pmf_3d_mult_cells_part2_gpu(const int ind, const double *dr,
                            const double *ds, const double *dt,
                            const double *in_r, const double *in_s,
                            const double *in_t, double *out) {
  if(!(ind < dg_np))
    return;
  const double *dr_mat = &dr[(p - 1) * DG_NP * DG_NP];
  const double *ds_mat = &ds[(p - 1) * DG_NP * DG_NP];
  const double *dt_mat = &dt[(p - 1) * DG_NP * DG_NP];

  double tmp = 0.0;
  for(int n = 0; n < dg_np; n++) {
    int mat_ind = DG_MAT_IND(n, ind, dg_np, dg_np);
    tmp += dr_mat[mat_ind] * in_r[n];
    tmp += ds_mat[mat_ind] * in_s[n];
    tmp += dt_mat[mat_ind] * in_t[n];
  }
  out[ind] += tmp;
}

// CUDA kernel function
template<int p, int NUM_CELLS>
__global__ void _op_cuda_pmf_3d_mult_cells_merged(
  const int *__restrict arg0,
  const double *arg1,
  const double *arg2,
  const double *arg3,
  const double *arg4,
  const double *arg5,
  const double *__restrict arg6,
  const double *__restrict arg7,
  const double *__restrict arg8,
  const double *__restrict arg9,
  const double *__restrict arg10,
  const double *__restrict arg11,
  const double *__restrict arg12,
  const double *__restrict arg13,
  const double *__restrict arg14,
  const double *__restrict arg15,
  const double *__restrict arg16,
  const double *__restrict arg17,
  const double *__restrict arg18,
  const double *__restrict arg19,
  const double *__restrict arg20,
  const double *__restrict arg21,
  const double *__restrict arg22,
  double *arg23,
  int   set_size ) {
  __shared__ double ux_shared[NUM_CELLS * DG_NP];
  __shared__ double uy_shared[NUM_CELLS * DG_NP];
  __shared__ double uz_shared[NUM_CELLS * DG_NP];
  __shared__ double tmp_x_shared[NUM_CELLS * DG_NP];
  __shared__ double tmp_y_shared[NUM_CELLS * DG_NP];
  __shared__ double tmp_z_shared[NUM_CELLS * DG_NP];

  //process set elements
  for (int n = threadIdx.x + blockIdx.x * blockDim.x; n - threadIdx.x < set_size * DG_NP; n += blockDim.x * gridDim.x){
    const int node_id = n % DG_NP;
    const int cell_id = n / DG_NP;
    const int local_cell_id = (n / DG_NP) - ((n - threadIdx.x) / DG_NP);
    const int start_ind = ((n - threadIdx.x) / DG_NP) * DG_NP;
    const int num_elem  = min((n - threadIdx.x + blockDim.x) / DG_NP, set_size) - ((n - threadIdx.x) / DG_NP) + 1;
    //user-supplied kernel call
    const int np  = (p + 1) * (p + 2) * (p + 3) / 6;
    const int npf = (p + 1) * (p + 2) / 2;
    if(n < set_size * DG_NP)
      _pmf_3d_mult_cells_part1_gpu<p,np,npf>(node_id,
                               arg2,
                               arg1,
                               arg15 + cell_id,
                               arg16 + cell_id * DG_NUM_FACES * DG_NPF,
                               arg17 + cell_id * DG_NUM_FACES * DG_NPF,
                               arg18 + cell_id * DG_NUM_FACES * DG_NPF,
                               arg19 + cell_id * DG_NUM_FACES * DG_NPF,
                               arg20 + cell_id * DG_NP,
                               arg21 + cell_id * DG_NP,
                               arg22 + cell_id * DG_NP,
                               tmp_x_shared + local_cell_id * DG_NP,
                               tmp_y_shared + local_cell_id * DG_NP,
                               tmp_z_shared + local_cell_id * DG_NP,
                               arg23 + cell_id * DG_NP);
    __syncthreads();
    for(int i = threadIdx.x; i < num_elem * DG_NP; i += blockDim.x) {
      int curr_cell = i / DG_NP + (n - threadIdx.x) / DG_NP;
      ux_shared[i] = *(arg6 + curr_cell) * tmp_x_shared[i] + *(arg9 + curr_cell) * tmp_y_shared[i] + *(arg12 + curr_cell) * tmp_z_shared[i];
      uy_shared[i] = *(arg7 + curr_cell) * tmp_x_shared[i] + *(arg10 + curr_cell) * tmp_y_shared[i] + *(arg13 + curr_cell) * tmp_z_shared[i];
      uz_shared[i] = *(arg8 + curr_cell) * tmp_x_shared[i] + *(arg11 + curr_cell) * tmp_y_shared[i] + *(arg14 + curr_cell) * tmp_z_shared[i];
    }
    __syncthreads();
    if(n < set_size * DG_NP)
      _pmf_3d_mult_cells_part2_gpu<p,np>(node_id, arg3, arg4, arg5,
                                  ux_shared + local_cell_id * DG_NP,
                                  uy_shared + local_cell_id * DG_NP,
                                  uz_shared + local_cell_id * DG_NP,
                                  arg23 + cell_id * DG_NP);
  }
}

#include "timing.h"
extern Timing *timer;

//host stub function
void custom_kernel_pmf_3d_mult_cells_merged(const int order, char const *name, op_set set,
  op_arg arg0,
  op_arg arg1,
  op_arg arg2,
  op_arg arg3,
  op_arg arg4,
  op_arg arg5,
  op_arg arg6,
  op_arg arg7,
  op_arg arg8,
  op_arg arg9,
  op_arg arg10,
  op_arg arg11,
  op_arg arg12,
  op_arg arg13,
  op_arg arg14,
  op_arg arg15,
  op_arg arg16,
  op_arg arg17,
  op_arg arg18,
  op_arg arg19,
  op_arg arg20,
  op_arg arg21,
  op_arg arg22,
  op_arg arg23){

  double*arg1h = (double *)arg1.data;
  double*arg2h = (double *)arg2.data;
  double*arg3h = (double *)arg3.data;
  double*arg4h = (double *)arg4.data;
  double*arg5h = (double *)arg5.data;
  int nargs = 24;
  op_arg args[24];

  args[0] = arg0;
  args[1] = arg1;
  args[2] = arg2;
  args[3] = arg3;
  args[4] = arg4;
  args[5] = arg5;
  args[6] = arg6;
  args[7] = arg7;
  args[8] = arg8;
  args[9] = arg9;
  args[10] = arg10;
  args[11] = arg11;
  args[12] = arg12;
  args[13] = arg13;
  args[14] = arg14;
  args[15] = arg15;
  args[16] = arg16;
  args[17] = arg17;
  args[18] = arg18;
  args[19] = arg19;
  args[20] = arg20;
  args[21] = arg21;
  args[22] = arg22;
  args[23] = arg23;

  if (OP_diags>2) {
    printf(" kernel routine w/o indirection:  pmf_3d_mult_cells_merged");
  }

  int set_size = op_mpi_halo_exchanges_grouped(set, nargs, args, 2);
  if (set_size > 0) {

    //transfer constants to GPU
    int consts_bytes = 0;
    consts_bytes += ROUND_UP(DG_ORDER * DG_NUM_FACES * DG_NPF * DG_NP *sizeof(double));
    consts_bytes += ROUND_UP(DG_ORDER * DG_NP * DG_NP *sizeof(double));
    consts_bytes += ROUND_UP(DG_ORDER * DG_NP * DG_NP *sizeof(double));
    consts_bytes += ROUND_UP(DG_ORDER * DG_NP * DG_NP *sizeof(double));
    consts_bytes += ROUND_UP(DG_ORDER * DG_NP * DG_NP *sizeof(double));
    reallocConstArrays(consts_bytes);
    consts_bytes = 0;
    arg1.data   = OP_consts_h + consts_bytes;
    arg1.data_d = OP_consts_d + consts_bytes;
    memcpy(arg1.data, arg1h, DG_ORDER * DG_NUM_FACES * DG_NPF * DG_NP * sizeof(double));
    consts_bytes += ROUND_UP(DG_ORDER * DG_NUM_FACES * DG_NPF * DG_NP * sizeof(double));
    arg2.data   = OP_consts_h + consts_bytes;
    arg2.data_d = OP_consts_d + consts_bytes;
    memcpy(arg2.data, arg2h, DG_ORDER * DG_NP * DG_NP * sizeof(double));
    consts_bytes += ROUND_UP(DG_ORDER * DG_NP * DG_NP * sizeof(double));
    arg3.data   = OP_consts_h + consts_bytes;
    arg3.data_d = OP_consts_d + consts_bytes;
    memcpy(arg3.data, arg3h, DG_ORDER * DG_NP * DG_NP * sizeof(double));
    consts_bytes += ROUND_UP(DG_ORDER * DG_NP * DG_NP * sizeof(double));
    arg4.data   = OP_consts_h + consts_bytes;
    arg4.data_d = OP_consts_d + consts_bytes;
    memcpy(arg4.data, arg4h, DG_ORDER * DG_NP * DG_NP * sizeof(double));
    consts_bytes += ROUND_UP(DG_ORDER * DG_NP * DG_NP * sizeof(double));
    arg5.data   = OP_consts_h + consts_bytes;
    arg5.data_d = OP_consts_d + consts_bytes;
    memcpy(arg5.data, arg5h, DG_ORDER * DG_NP * DG_NP * sizeof(double));
    consts_bytes += ROUND_UP(DG_ORDER * DG_NP * DG_NP * sizeof(double));
    mvConstArraysToDevice(consts_bytes);

    //set CUDA execution parameters
    const int nthread = (256 /  DG_NP) * DG_NP;
    const int nblocks = 200 < (set->size * DG_NP) / nthread + 1 ? 200 : (set->size * DG_NP) / nthread + 1;
    const int num_cells = (nthread / DG_NP) + 1;

    switch(order) {
      case 1:
        _op_cuda_pmf_3d_mult_cells_merged<1,num_cells><<<nblocks,nthread>>>(
          (int *) arg0.data_d,
          (double *) arg1.data_d,
          (double *) arg2.data_d,
          (double *) arg3.data_d,
          (double *) arg4.data_d,
          (double *) arg5.data_d,
          (double *) arg6.data_d,
          (double *) arg7.data_d,
          (double *) arg8.data_d,
          (double *) arg9.data_d,
          (double *) arg10.data_d,
          (double *) arg11.data_d,
          (double *) arg12.data_d,
          (double *) arg13.data_d,
          (double *) arg14.data_d,
          (double *) arg15.data_d,
          (double *) arg16.data_d,
          (double *) arg17.data_d,
          (double *) arg18.data_d,
          (double *) arg19.data_d,
          (double *) arg20.data_d,
          (double *) arg21.data_d,
          (double *) arg22.data_d,
          (double *) arg23.data_d,
          set->size );
        break;
      case 2:
        _op_cuda_pmf_3d_mult_cells_merged<2,num_cells><<<nblocks,nthread>>>(
          (int *) arg0.data_d,
          (double *) arg1.data_d,
          (double *) arg2.data_d,
          (double *) arg3.data_d,
          (double *) arg4.data_d,
          (double *) arg5.data_d,
          (double *) arg6.data_d,
          (double *) arg7.data_d,
          (double *) arg8.data_d,
          (double *) arg9.data_d,
          (double *) arg10.data_d,
          (double *) arg11.data_d,
          (double *) arg12.data_d,
          (double *) arg13.data_d,
          (double *) arg14.data_d,
          (double *) arg15.data_d,
          (double *) arg16.data_d,
          (double *) arg17.data_d,
          (double *) arg18.data_d,
          (double *) arg19.data_d,
          (double *) arg20.data_d,
          (double *) arg21.data_d,
          (double *) arg22.data_d,
          (double *) arg23.data_d,
          set->size );
        break;
      case 3:
        timer->startTimer("fpmf_cells_merged 3rd order");
        _op_cuda_pmf_3d_mult_cells_merged<3,num_cells><<<nblocks,nthread>>>(
          (int *) arg0.data_d,
          (double *) arg1.data_d,
          (double *) arg2.data_d,
          (double *) arg3.data_d,
          (double *) arg4.data_d,
          (double *) arg5.data_d,
          (double *) arg6.data_d,
          (double *) arg7.data_d,
          (double *) arg8.data_d,
          (double *) arg9.data_d,
          (double *) arg10.data_d,
          (double *) arg11.data_d,
          (double *) arg12.data_d,
          (double *) arg13.data_d,
          (double *) arg14.data_d,
          (double *) arg15.data_d,
          (double *) arg16.data_d,
          (double *) arg17.data_d,
          (double *) arg18.data_d,
          (double *) arg19.data_d,
          (double *) arg20.data_d,
          (double *) arg21.data_d,
          (double *) arg22.data_d,
          (double *) arg23.data_d,
          set->size );
        cutilSafeCall(hipDeviceSynchronize());
        timer->endTimer("fpmf_cells_merged 3rd order");
        break;
      case 4:
        _op_cuda_pmf_3d_mult_cells_merged<4,num_cells><<<nblocks,nthread>>>(
          (int *) arg0.data_d,
          (double *) arg1.data_d,
          (double *) arg2.data_d,
          (double *) arg3.data_d,
          (double *) arg4.data_d,
          (double *) arg5.data_d,
          (double *) arg6.data_d,
          (double *) arg7.data_d,
          (double *) arg8.data_d,
          (double *) arg9.data_d,
          (double *) arg10.data_d,
          (double *) arg11.data_d,
          (double *) arg12.data_d,
          (double *) arg13.data_d,
          (double *) arg14.data_d,
          (double *) arg15.data_d,
          (double *) arg16.data_d,
          (double *) arg17.data_d,
          (double *) arg18.data_d,
          (double *) arg19.data_d,
          (double *) arg20.data_d,
          (double *) arg21.data_d,
          (double *) arg22.data_d,
          (double *) arg23.data_d,
          set->size );
        break;
      case 5:
        _op_cuda_pmf_3d_mult_cells_merged<5,num_cells><<<nblocks,nthread>>>(
          (int *) arg0.data_d,
          (double *) arg1.data_d,
          (double *) arg2.data_d,
          (double *) arg3.data_d,
          (double *) arg4.data_d,
          (double *) arg5.data_d,
          (double *) arg6.data_d,
          (double *) arg7.data_d,
          (double *) arg8.data_d,
          (double *) arg9.data_d,
          (double *) arg10.data_d,
          (double *) arg11.data_d,
          (double *) arg12.data_d,
          (double *) arg13.data_d,
          (double *) arg14.data_d,
          (double *) arg15.data_d,
          (double *) arg16.data_d,
          (double *) arg17.data_d,
          (double *) arg18.data_d,
          (double *) arg19.data_d,
          (double *) arg20.data_d,
          (double *) arg21.data_d,
          (double *) arg22.data_d,
          (double *) arg23.data_d,
          set->size );
        break;
    }
  }
  op_mpi_set_dirtybit_cuda(nargs, args);
  cutilSafeCall(hipDeviceSynchronize());
}
