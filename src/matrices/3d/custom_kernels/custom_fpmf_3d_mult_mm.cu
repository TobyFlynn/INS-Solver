#include "hip/hip_runtime.h"
#include "op_lib_cpp.h"
#include "op_cuda_rt_support.h"
#include "op_cuda_reduction.h"

#include "dg_compiler_defs.h"

template<int p, int dg_np>
__device__ void _fpmf_3d_mult_mm_gpu(const int node, const double *mass,
                                    const double *J, const double *mm_factor,
                                    const double *in, double *out) {
  const double *mass_mat = &mass[(p - 1) * 20 * 20];
  DG_FP tmp = 0.0;
  for(int n = 0; n < dg_np; n++) {
    int ind = DG_MAT_IND(node, n, dg_np, dg_np);
    tmp += mm_factor[n] * mass_mat[ind] * in[n];
  }
  out[node] += tmp * J[0];
}

// CUDA kernel function
template<int p>
__global__ void _op_cuda_fpmf_3d_mult_mm(
  const int *__restrict arg0,
  const double *arg1,
  const double *__restrict arg2,
  const double *__restrict arg3,
  const double *__restrict arg4,
  double *arg5,
  int   set_size ) {

  const int np = (p + 1) * (p + 2) * (p + 3) / 6;
  //process set elements
  for(int n = threadIdx.x + blockIdx.x * blockDim.x;
      n < set_size * np; n += blockDim.x * gridDim.x){

    //user-supplied kernel call
    const int node = n % np;
    const int cell = n / np;
    _fpmf_3d_mult_mm_gpu<p,np>(node,
                    arg1,
                    arg2+cell*1,
                    arg3+cell*DG_NP,
                    arg4+cell*DG_NP,
                    arg5+cell*DG_NP);
  }
}


//host stub function
void custom_kernel_fpmf_3d_mult_mm(const int order, char const *name, op_set set,
  op_arg arg0,
  op_arg arg1,
  op_arg arg2,
  op_arg arg3,
  op_arg arg4,
  op_arg arg5){

  double*arg1h = (double *)arg1.data;
  int nargs = 6;
  op_arg args[6];

  args[0] = arg0;
  args[1] = arg1;
  args[2] = arg2;
  args[3] = arg3;
  args[4] = arg4;
  args[5] = arg5;

  if (OP_diags>2) {
    printf(" kernel routine w/o indirection:  fpmf_3d_mult_mm");
  }

  int set_size = op_mpi_halo_exchanges_grouped(set, nargs, args, 2);
  if (set_size > 0) {

    //transfer constants to GPU
    int consts_bytes = 0;
    consts_bytes += ROUND_UP(1200*sizeof(double));
    reallocConstArrays(consts_bytes);
    consts_bytes = 0;
    arg1.data   = OP_consts_h + consts_bytes;
    arg1.data_d = OP_consts_d + consts_bytes;
    memcpy(arg1.data, arg1h, 1200*sizeof(double));
    consts_bytes += ROUND_UP(1200*sizeof(double));
    mvConstArraysToDevice(consts_bytes);

    //set CUDA execution parameters
    const int nthread = 256;
    const int nblocks = 200 < (set->size * DG_NP) / nthread + 1 ? 200 : (set->size * DG_NP) / nthread + 1;

    switch(order) {
      case 1:
        _op_cuda_fpmf_3d_mult_mm<1><<<nblocks,nthread>>>(
          (int *) arg0.data_d,
          (double *) arg1.data_d,
          (double *) arg2.data_d,
          (double *) arg3.data_d,
          (double *) arg4.data_d,
          (double *) arg5.data_d,
          set->size );
        break;
      case 2:
        _op_cuda_fpmf_3d_mult_mm<2><<<nblocks,nthread>>>(
          (int *) arg0.data_d,
          (double *) arg1.data_d,
          (double *) arg2.data_d,
          (double *) arg3.data_d,
          (double *) arg4.data_d,
          (double *) arg5.data_d,
          set->size );
        break;
      case 3:
        _op_cuda_fpmf_3d_mult_mm<3><<<nblocks,nthread>>>(
          (int *) arg0.data_d,
          (double *) arg1.data_d,
          (double *) arg2.data_d,
          (double *) arg3.data_d,
          (double *) arg4.data_d,
          (double *) arg5.data_d,
          set->size );
        break;
      case 4:
        _op_cuda_fpmf_3d_mult_mm<4><<<nblocks,nthread>>>(
          (int *) arg0.data_d,
          (double *) arg1.data_d,
          (double *) arg2.data_d,
          (double *) arg3.data_d,
          (double *) arg4.data_d,
          (double *) arg5.data_d,
          set->size );
        break;
      case 5:
        _op_cuda_fpmf_3d_mult_mm<5><<<nblocks,nthread>>>(
          (int *) arg0.data_d,
          (double *) arg1.data_d,
          (double *) arg2.data_d,
          (double *) arg3.data_d,
          (double *) arg4.data_d,
          (double *) arg5.data_d,
          set->size );
        break;
    }
  }
  op_mpi_set_dirtybit_cuda(nargs, args);
  cutilSafeCall(hipDeviceSynchronize());
}
