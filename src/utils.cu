#include "op_seq.h"

#include <memory>

#include "dg_utils.h"

double *getOP2PtrDevice(op_dat dat, op_access acc) {
  op_arg args[] = {
    op_arg_dat(dat, -1, OP_ID, dat->dim, "double", acc)
  };
  op_mpi_halo_exchanges_cuda(dat->set, 1, args);
  op_mpi_wait_all(1, args);
  return (double *) dat->data_d;
}

void releaseOP2PtrDevice(op_dat dat, op_access acc, const double *ptr) {
  op_arg args[] = {
    op_arg_dat(dat, -1, OP_ID, dat->dim, "double", acc)
  };
  op_mpi_set_dirtybit_cuda(1, args);

  ptr = nullptr;
}

double *getOP2PtrHost(op_dat dat, op_access acc) {
  op_arg args[] = {
    op_arg_dat(dat, -1, OP_ID, dat->dim, "double", acc)
  };
  op_mpi_halo_exchanges_cuda(dat->set, 1, args);
  op_mpi_wait_all(1, args);
  double *res = (double *)malloc(dat->set->size * dat->dim * sizeof(double));
  hipMemcpy(res, dat->data_d, dat->set->size * dat->dim * sizeof(double), hipMemcpyDeviceToHost);
  return res;
}

void releaseOP2PtrHost(op_dat dat, op_access acc, const double *ptr) {
  op_arg args[] = {
    op_arg_dat(dat, -1, OP_ID, dat->dim, "double", acc)
  };

  if(acc != OP_READ) {
    hipMemcpy(dat->data_d, ptr, dat->set->size * dat->dim * sizeof(double), hipMemcpyHostToDevice);
  }

  op_mpi_set_dirtybit_cuda(1, args);

  free((void *)ptr);
  ptr = nullptr;
}

double *getOP2PtrDeviceMap(op_dat dat, op_map map, op_access acc) {
  op_arg args[] = {
    op_arg_dat(dat, 0, map, dat->dim, "double", acc),
    op_arg_dat(dat, 1, map, dat->dim, "double", acc)
  };
  op_mpi_halo_exchanges_cuda(map->from, 2, args);
  op_mpi_wait_all(2, args);
  
  return (double *) dat->data_d;
}

void releaseOP2PtrDeviceMap(op_dat dat, op_map map, op_access acc, const double *ptr) {
  op_arg args[] = {
    op_arg_dat(dat, 0, map, dat->dim, "double", acc),
    op_arg_dat(dat, 1, map, dat->dim, "double", acc)
  };
  op_mpi_set_dirtybit_cuda(2, args);

  ptr = nullptr;
}

double *getOP2PtrHostMap(op_dat dat, op_map map, op_access acc) {
  op_arg args[] = {
    op_arg_dat(dat, 0, map, dat->dim, "double", acc),
    op_arg_dat(dat, 1, map, dat->dim, "double", acc)
  };
  op_mpi_halo_exchanges_cuda(map->from, 2, args);
  op_mpi_wait_all(2, args);

  // TODO check that this shouldn't be more than dat->set->size
  double *res = (double *)malloc(dat->set->size * dat->dim * sizeof(double));
  hipMemcpy(res, dat->data_d, dat->set->size * dat->dim * sizeof(double), hipMemcpyDeviceToHost);
  return res;
}

void releaseOP2PtrHostMap(op_dat dat, op_map map, op_access acc, const double *ptr) {
  op_arg args[] = {
    op_arg_dat(dat, 0, map, dat->dim, "double", acc),
    op_arg_dat(dat, 1, map, dat->dim, "double", acc)
  };

  if(acc != OP_READ) {
    hipMemcpy(dat->data_d, ptr, dat->set->size * dat->dim * sizeof(double), hipMemcpyHostToDevice);
  }

  op_mpi_set_dirtybit(2, args);
  free((void *)ptr);
  ptr = nullptr;
}

double *getOP2Array(op_dat dat) {
  op_arg args[] = {
    op_arg_dat(dat, -1, OP_ID, DG_SUB_CELLS, "double", OP_READ)
  };
  op_mpi_halo_exchanges_cuda(dat->set, 1, args);
  double *res = (double *)malloc(dat->set->size * DG_SUB_CELLS * sizeof(double));
  hipMemcpy(res, dat->data_d, dat->set->size * DG_SUB_CELLS * sizeof(double), hipMemcpyDeviceToHost);
  op_mpi_set_dirtybit_cuda(1, args);
  return res;
}
