#include "op_seq.h"

#ifdef INS_MPI
#include "op_lib_mpi.h"
#endif

#include <memory>

#include "dg_utils.h"

double *getOP2PtrDevice(op_dat dat, op_access acc) {
  op_arg args[] = {
    op_arg_dat(dat, -1, OP_ID, dat->dim, "double", acc)
  };
  op_mpi_halo_exchanges_cuda(dat->set, 1, args);
  op_mpi_wait_all(1, args);
  return (double *) dat->data_d;
}

void releaseOP2PtrDevice(op_dat dat, op_access acc, const double *ptr) {
  op_arg args[] = {
    op_arg_dat(dat, -1, OP_ID, dat->dim, "double", acc)
  };
  op_mpi_set_dirtybit_cuda(1, args);

  ptr = nullptr;
}

double *getOP2PtrHost(op_dat dat, op_access acc) {
  op_arg args[] = {
    op_arg_dat(dat, -1, OP_ID, dat->dim, "double", acc)
  };
  op_mpi_halo_exchanges_cuda(dat->set, 1, args);
  op_mpi_wait_all(1, args);
  double *res = (double *)malloc(dat->set->size * dat->dim * sizeof(double));
  hipMemcpy(res, dat->data_d, dat->set->size * dat->dim * sizeof(double), hipMemcpyDeviceToHost);
  return res;
}

void releaseOP2PtrHost(op_dat dat, op_access acc, const double *ptr) {
  op_arg args[] = {
    op_arg_dat(dat, -1, OP_ID, dat->dim, "double", acc)
  };

  if(acc != OP_READ) {
    hipMemcpy(dat->data_d, ptr, dat->set->size * dat->dim * sizeof(double), hipMemcpyHostToDevice);
  }

  op_mpi_set_dirtybit_cuda(1, args);

  free((void *)ptr);
  ptr = nullptr;
}

double *getOP2PtrDeviceMap(op_dat dat, op_map map, op_access acc) {
  op_arg args[] = {
    op_arg_dat(dat, 0, map, dat->dim, "double", acc),
    op_arg_dat(dat, 1, map, dat->dim, "double", acc)
  };
  op_mpi_halo_exchanges_cuda(map->from, 2, args);
  op_mpi_wait_all(2, args);
  
  return (double *) dat->data_d;
}

void releaseOP2PtrDeviceMap(op_dat dat, op_map map, op_access acc, const double *ptr) {
  op_arg args[] = {
    op_arg_dat(dat, 0, map, dat->dim, "double", acc),
    op_arg_dat(dat, 1, map, dat->dim, "double", acc)
  };
  op_mpi_set_dirtybit_cuda(2, args);

  ptr = nullptr;
}

double *getOP2PtrHostMap(op_dat dat, op_map map, op_access acc) {    
  op_arg args[] = {
    op_arg_dat(dat, 0, map, dat->dim, "double", acc),
    op_arg_dat(dat, 1, map, dat->dim, "double", acc)
  };
  op_mpi_halo_exchanges_grouped(map->from, 2, args, 2);
  op_mpi_wait_all_grouped(2, args, 2);

  #ifdef INS_MPI
  double *res = (double *)malloc(dat->dim * (dat->set->size + OP_import_exec_list[dat->set->index]->size + OP_import_nonexec_list[dat->set->index]->size) * sizeof(double));
  hipMemcpy(res, dat->data_d, (dat->set->size + OP_import_exec_list[dat->set->index]->size + OP_import_nonexec_list[dat->set->index]->size) * dat->dim * sizeof(double), hipMemcpyDeviceToHost); 
  #else
  double *res = (double *)malloc(dat->set->size * dat->dim * sizeof(double));
  hipMemcpy(res, dat->data_d, dat->set->size * dat->dim * sizeof(double), hipMemcpyDeviceToHost);
  #endif
  return res;
}

void releaseOP2PtrHostMap(op_dat dat, op_map map, op_access acc, const double *ptr) { 
  op_arg args[] = {
    op_arg_dat(dat, 0, map, dat->dim, "double", acc),
    op_arg_dat(dat, 1, map, dat->dim, "double", acc)
  };

  if(acc != OP_READ) {
    #ifdef INS_MPI
    hipMemcpy(dat->data_d, ptr, (dat->set->size + OP_import_exec_list[dat->set->index]->size + OP_import_nonexec_list[dat->set->index]->size) * dat->dim * sizeof(double), hipMemcpyHostToDevice);
    #else
    hipMemcpy(dat->data_d, ptr, dat->set->size * dat->dim * sizeof(double), hipMemcpyHostToDevice);
    #endif
  }

  op_mpi_set_dirtybit_cuda(2, args);

  free((void *)ptr);
  ptr = nullptr;
}

double *getOP2Array(op_dat dat) {
  op_arg args[] = {
    op_arg_dat(dat, -1, OP_ID, DG_SUB_CELLS, "double", OP_READ)
  };
  op_mpi_halo_exchanges_cuda(dat->set, 1, args);
  double *res = (double *)malloc(dat->set->size * DG_SUB_CELLS * sizeof(double));
  hipMemcpy(res, dat->data_d, dat->set->size * DG_SUB_CELLS * sizeof(double), hipMemcpyDeviceToHost);
  op_mpi_set_dirtybit_cuda(1, args);
  return res;
}
