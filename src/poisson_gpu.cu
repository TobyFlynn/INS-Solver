#include "poisson.h"

#ifdef INS_MPI
#include "mpi_helper_func.h"
#endif

#include "dg_utils.h"

int PoissonSolve::get_local_unknowns() {
  op_arg op2_args[] = {
    op_arg_dat(mesh->order, -1, OP_ID, 1, "int", OP_READ)
  };
  op_mpi_halo_exchanges_cuda(mesh->order->set, 1, op2_args);
  const int setSize = mesh->order->set->size;
  int *tempOrder = (int *)malloc(setSize * sizeof(int));
  hipMemcpy(tempOrder, mesh->order->data_d, setSize * sizeof(int), hipMemcpyDeviceToHost);
  int local_unkowns = 0;
  for(int i = 0; i < setSize; i++) {
    int Np, Nfp;
    DGUtils::basic_constants(tempOrder[i], &Np, &Nfp);
    local_unkowns += Np;
  }
  free(tempOrder);
  op_mpi_set_dirtybit_cuda(1, op2_args);
  return local_unkowns;
}

// Copy PETSc vec array to OP2 dat
void PoissonSolve::copy_vec_to_dat(op_dat dat, const double *dat_d) {
  op_arg copy_args[] = {
    op_arg_dat(dat, -1, OP_ID, DG_NP, "double", OP_WRITE),
    op_arg_dat(mesh->order, -1, OP_ID, 1, "int", OP_READ)
  };
  op_mpi_halo_exchanges_cuda(dat->set, 2, copy_args);

  int setSize = dat->set->size;
  int *tempOrder = (int *)malloc(setSize * sizeof(int));
  hipMemcpy(tempOrder, mesh->order->data_d, setSize * sizeof(int), hipMemcpyDeviceToHost);

  int vec_ind = 0;
  int block_start = 0;
  int block_count = 0;
  for(int i = 0; i < setSize; i++) {
    const int N = tempOrder[i];

    if(N == DG_ORDER) {
      if(block_count == 0) {
        block_start = i;
        block_count++;
        continue;
      } else {
        block_count++;
        continue;
      }
    } else {
      if(block_count != 0) {
        double *block_start_dat_c = (double *)dat->data_d + block_start * dat->dim;
        hipMemcpy(block_start_dat_c, dat_d + vec_ind, block_count * DG_NP * sizeof(double), hipMemcpyDeviceToDevice);
        vec_ind += DG_NP * block_count;
      }
      block_count = 0;
    }

    double *v_c = (double *)dat->data_d + i * dat->dim;
    int Np, Nfp;
    DGUtils::basic_constants(N, &Np, &Nfp);

    hipMemcpy(v_c, dat_d + vec_ind, Np * sizeof(double), hipMemcpyDeviceToDevice);
    vec_ind += Np;
  }

  if(block_count != 0) {
    double *block_start_dat_c = (double *)dat->data_d + block_start * dat->dim;
    hipMemcpy(block_start_dat_c, dat_d + vec_ind, block_count * DG_NP * sizeof(double), hipMemcpyDeviceToDevice);
    vec_ind += DG_NP * block_count;
  }
  free(tempOrder);
  op_mpi_set_dirtybit_cuda(2, copy_args);
}

// Copy OP2 dat to PETSc vec array
void PoissonSolve::copy_dat_to_vec(op_dat dat, double *dat_d) {
  op_arg copy_args[] = {
    op_arg_dat(dat, -1, OP_ID, DG_NP, "double", OP_READ),
    op_arg_dat(mesh->order, -1, OP_ID, 1, "int", OP_READ)
  };
  op_mpi_halo_exchanges_cuda(dat->set, 2, copy_args);

  int setSize = dat->set->size;
  int *tempOrder = (int *)malloc(setSize * sizeof(int));
  hipMemcpy(tempOrder, mesh->order->data_d, setSize * sizeof(int), hipMemcpyDeviceToHost);

  int vec_ind = 0;
  int block_start = 0;
  int block_count = 0;
  for(int i = 0; i < setSize; i++) {
    const int N = tempOrder[i];

    if(N == DG_ORDER) {
      if(block_count == 0) {
        block_start = i;
        block_count++;
        continue;
      } else {
        block_count++;
        continue;
      }
    } else {
      if(block_count != 0) {
        const double *block_start_dat_c = (double *)dat->data_d + block_start * dat->dim;
        hipMemcpy(dat_d + vec_ind, block_start_dat_c, block_count * DG_NP * sizeof(double), hipMemcpyDeviceToDevice);
        vec_ind += DG_NP * block_count;
      }
      block_count = 0;
    }

    const double *v_c = (double *)dat->data_d + i * dat->dim;
    int Np, Nfp;
    DGUtils::basic_constants(N, &Np, &Nfp);

    hipMemcpy(dat_d + vec_ind, v_c, Np * sizeof(double), hipMemcpyDeviceToDevice);
    vec_ind += Np;
  }

  if(block_count != 0) {
    const double *block_start_dat_c = (double *)dat->data_d + block_start * dat->dim;
    hipMemcpy(dat_d + vec_ind, block_start_dat_c, block_count * DG_NP * sizeof(double), hipMemcpyDeviceToDevice);
    vec_ind += DG_NP * block_count;
  }
  free(tempOrder);
  op_mpi_set_dirtybit_cuda(2, copy_args);
}

// Create a PETSc vector for GPUs
void PoissonSolve::create_vec(Vec *v) {
  VecCreate(PETSC_COMM_WORLD, v);
  VecSetType(*v, VECCUDA);
  VecSetSizes(*v, unknowns, PETSC_DECIDE);
}

// Destroy a PETSc vector
void PoissonSolve::destroy_vec(Vec *v) {
  VecDestroy(v);
}

// Load a PETSc vector with values from an OP2 dat for GPUs
void PoissonSolve::load_vec(Vec *v, op_dat v_dat) {
  double *v_ptr;
  VecCUDAGetArray(*v, &v_ptr);

  copy_dat_to_vec(v_dat, v_ptr);

  VecCUDARestoreArray(*v, &v_ptr);
}

// Load an OP2 dat with the values from a PETSc vector for GPUs
void PoissonSolve::store_vec(Vec *v, op_dat v_dat) {
  const double *v_ptr;
  VecCUDAGetArrayRead(*v, &v_ptr);

  copy_vec_to_dat(v_dat, v_ptr);

  VecCUDARestoreArrayRead(*v, &v_ptr);
}

PetscErrorCode matAMult(Mat A, Vec x, Vec y) {
  PoissonSolve *poisson;
  MatShellGetContext(A, &poisson);
  const double *x_ptr;
  double *y_ptr;
  VecCUDAGetArrayRead(x, &x_ptr);
  VecCUDAGetArray(y, &y_ptr);

  poisson->calc_rhs(x_ptr, y_ptr);

  VecCUDARestoreArrayRead(x, &x_ptr);
  VecCUDARestoreArray(y, &y_ptr);
  return 0;
}

void PoissonSolve::create_shell_mat(Mat *m) {
  MatCreateShell(PETSC_COMM_WORLD, unknowns, unknowns, PETSC_DETERMINE, PETSC_DETERMINE, this, m);
  MatShellSetOperation(*m, MATOP_MULT, (void(*)(void))matAMult);
  MatShellSetVecType(*m, VECCUDA);
}

PetscErrorCode precon(PC pc, Vec x, Vec y) {
  PoissonSolve *poisson;
  PCShellGetContext(pc, (void **)&poisson);
  const double *x_ptr;
  double *y_ptr;
  VecCUDAGetArrayRead(x, &x_ptr);
  VecCUDAGetArray(y, &y_ptr);

  poisson->precond(x_ptr, y_ptr);

  VecCUDARestoreArrayRead(x, &x_ptr);
  VecCUDARestoreArray(y, &y_ptr);
  return 0;
}

void PoissonSolve::set_shell_pc(PC pc) {
  PCShellSetApply(pc, precon);
  PCShellSetContext(pc, this);
}

void PoissonSolve::setGlbInd() {
  int global_ind = 0;
  #ifdef INS_MPI
  global_ind = get_global_mat_start_ind(unknowns);
  #endif
  op_arg args[] = {
    op_arg_dat(mesh->order, -1, OP_ID, 1, "int", OP_READ),
    op_arg_dat(glb_ind, -1, OP_ID, 1, "int", OP_WRITE)
  };
  op_mpi_halo_exchanges_cuda(mesh->cells, 2, args);

  const int setSize = mesh->cells->size;
  int *tempOrder = (int *)malloc(setSize * sizeof(int));
  hipMemcpy(tempOrder, mesh->order->data_d, setSize * sizeof(int), hipMemcpyDeviceToHost);
  int *data_ptr = (int *)malloc(setSize * sizeof(int));
  hipMemcpy(data_ptr, glb_ind->data_d, setSize * sizeof(int), hipMemcpyDeviceToHost);

  int ind = global_ind;
  for(int i = 0; i < mesh->cells->size; i++) {
    int Np, Nfp;
    DGUtils::basic_constants(tempOrder[i], &Np, &Nfp);
    data_ptr[i] = ind;
    ind += Np;
  }

  hipMemcpy(glb_ind->data_d, data_ptr, setSize * sizeof(int), hipMemcpyHostToDevice);

  op_mpi_set_dirtybit_cuda(2, args);
  free(data_ptr);
  free(tempOrder);
}

void PoissonSolve::setMatrix() {
  if(pMatInit) {
    MatDestroy(&pMat);
  }
  MatCreate(PETSC_COMM_WORLD, &pMat);
  pMatInit = true;
  MatSetSizes(pMat, unknowns, unknowns, PETSC_DECIDE, PETSC_DECIDE);

  #ifdef INS_MPI
  MatSetType(pMat, MATMPIAIJCUSPARSE);
  MatMPIAIJSetPreallocation(pMat, DG_NP * 4, NULL, 0, NULL);
  #else
  MatSetType(pMat, MATSEQAIJCUSPARSE);
  MatSeqAIJSetPreallocation(pMat, DG_NP * 4, NULL);
  #endif
  MatSetOption(pMat, MAT_NEW_NONZERO_ALLOCATION_ERR, PETSC_FALSE);

  // Add cubature OP to Poisson matrix
  op_arg args[] = {
    op_arg_dat(op1, -1, OP_ID, DG_NP * DG_NP, "double", OP_READ),
    op_arg_dat(glb_ind, -1, OP_ID, 1, "int", OP_READ),
    op_arg_dat(mesh->order, -1, OP_ID, 1, "int", OP_READ)
  };
  op_mpi_halo_exchanges_cuda(mesh->cells, 3, args);

  const int setSize = mesh->cells->size;
  double *op1_data = (double *)malloc(DG_NP * DG_NP * setSize * sizeof(double));
  int *glb   = (int *)malloc(setSize * sizeof(int));
  int *order = (int *)malloc(setSize * sizeof(int));
  hipMemcpy(op1_data, op1->data_d, setSize * DG_NP * DG_NP * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(glb, glb_ind->data_d, setSize * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(order, mesh->order->data_d, setSize * sizeof(int), hipMemcpyDeviceToHost);
  op_mpi_set_dirtybit_cuda(3, args);

  for(int i = 0; i < setSize; i++) {
    int Np, Nfp;
    DGUtils::basic_constants(order[i], &Np, &Nfp);
    int currentRow = glb[i];
    int currentCol = glb[i];

    // Convert data to row major format
    for(int m = 0; m < Np; m++) {
      for(int n = 0; n < Np; n++) {
        int row = currentRow + m;
        int col = currentCol + n;
        double val = op1_data[i * DG_NP * DG_NP + m + n * Np];
        MatSetValues(pMat, 1, &row, 1, &col, &val, INSERT_VALUES);
      }
    }
  }

  free(op1_data);
  free(glb);
  free(order);

  op_arg edge_args[] = {
    op_arg_dat(op2[0], -1, OP_ID, DG_NP * DG_NP, "double", OP_READ),
    op_arg_dat(op2[1], -1, OP_ID, DG_NP * DG_NP, "double", OP_READ),
    op_arg_dat(glb_indL, -1, OP_ID, 1, "int", OP_READ),
    op_arg_dat(glb_indR, -1, OP_ID, 1, "int", OP_READ),
    op_arg_dat(orderL, -1, OP_ID, 1, "int", OP_READ),
    op_arg_dat(orderR, -1, OP_ID, 1, "int", OP_READ)
  };
  op_mpi_halo_exchanges_cuda(mesh->edges, 6, edge_args);
  double *op2L_data = (double *)malloc(DG_NP * DG_NP * mesh->edges->size * sizeof(double));
  double *op2R_data = (double *)malloc(DG_NP * DG_NP * mesh->edges->size * sizeof(double));
  int *glb_l = (int *)malloc(mesh->edges->size * sizeof(int));
  int *glb_r = (int *)malloc(mesh->edges->size * sizeof(int));
  int *order_l = (int *)malloc(mesh->edges->size * sizeof(int));
  int *order_r = (int *)malloc(mesh->edges->size * sizeof(int));

  hipMemcpy(op2L_data, op2[0]->data_d, DG_NP * DG_NP * mesh->edges->size * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(op2R_data, op2[1]->data_d, DG_NP * DG_NP * mesh->edges->size * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(glb_l, glb_indL->data_d, mesh->edges->size * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(glb_r, glb_indR->data_d, mesh->edges->size * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(order_l, orderL->data_d, mesh->edges->size * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(order_r, orderR->data_d, mesh->edges->size * sizeof(int), hipMemcpyDeviceToHost);

  // Add Gauss OP and OPf to Poisson matrix
  for(int i = 0; i < mesh->edges->size; i++) {
    int leftRow = glb_l[i];
    int rightRow = glb_r[i];
    int NpL, NpR, Nfp;
    DGUtils::basic_constants(order_l[i], &NpL, &Nfp);
    DGUtils::basic_constants(order_r[i], &NpR, &Nfp);

    // Gauss OPf
    // Convert data to row major format
    for(int m = 0; m < NpL; m++) {
      for(int n = 0; n < NpR; n++) {
        int row = leftRow + m;
        int col = rightRow + n;
        double val = op2L_data[i * DG_NP * DG_NP + m + n * NpL];
        MatSetValues(pMat, 1, &row, 1, &col, &val, INSERT_VALUES);
      }
    }
    // Convert data to row major format
    for(int m = 0; m < NpR; m++) {
      for(int n = 0; n < NpL; n++) {
        int row = rightRow + m;
        int col = leftRow + n;
        double val = op2R_data[i * DG_NP * DG_NP + m + n * NpR];
        MatSetValues(pMat, 1, &row, 1, &col, &val, INSERT_VALUES);
      }
    }
  }

  free(op2L_data);
  free(op2R_data);
  free(glb_l);
  free(glb_r);
  free(order_l);
  free(order_r);

  op_mpi_set_dirtybit_cuda(6, edge_args);

  MatAssemblyBegin(pMat, MAT_FINAL_ASSEMBLY);
  MatAssemblyEnd(pMat, MAT_FINAL_ASSEMBLY);
}
