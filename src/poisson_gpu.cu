#include "poisson.h"
#include "poisson_HYPRE.h"

#ifdef INS_MPI
#include "mpi_helper_func.h"
#endif

#include "dg_utils.h"

PetscErrorCode matAMult(Mat A, Vec x, Vec y) {
  PoissonSolve *poisson;
  MatShellGetContext(A, &poisson);
  const double *x_ptr;
  double *y_ptr;
  VecCUDAGetArrayRead(x, &x_ptr);
  VecCUDAGetArray(y, &y_ptr);

  poisson->calc_rhs(x_ptr, y_ptr);

  VecCUDARestoreArrayRead(x, &x_ptr);
  VecCUDARestoreArray(y, &y_ptr);
  return 0;
}

void PoissonSolve::create_shell_mat() {
  if(pMatInit)
    MatDestroy(&pMat);

  MatCreateShell(PETSC_COMM_WORLD, unknowns, unknowns, PETSC_DETERMINE, PETSC_DETERMINE, this, &pMat);
  MatShellSetOperation(pMat, MATOP_MULT, (void(*)(void))matAMult);
  MatShellSetVecType(pMat, VECCUDA);

  pMatInit = true;
}

PetscErrorCode precon(PC pc, Vec x, Vec y) {
  PoissonSolve *poisson;
  PCShellGetContext(pc, (void **)&poisson);
  const double *x_ptr;
  double *y_ptr;
  VecCUDAGetArrayRead(x, &x_ptr);
  VecCUDAGetArray(y, &y_ptr);

  poisson->precond(x_ptr, y_ptr);

  VecCUDARestoreArrayRead(x, &x_ptr);
  VecCUDARestoreArray(y, &y_ptr);
  return 0;
}

void PoissonSolve::set_shell_pc(PC pc) {
  PCShellSetApply(pc, precon);
  PCShellSetContext(pc, this);
}

void PoissonSolve::setGlbInd() {
  int global_ind = 0;
  #ifdef INS_MPI
  global_ind = get_global_mat_start_ind(unknowns);
  #endif
  op_arg args[] = {
    op_arg_dat(mesh->order, -1, OP_ID, 1, "int", OP_READ),
    op_arg_dat(glb_ind, -1, OP_ID, 1, "int", OP_WRITE)
  };
  op_mpi_halo_exchanges_cuda(mesh->cells, 2, args);

  const int setSize = mesh->cells->size;
  int *tempOrder = (int *)malloc(setSize * sizeof(int));
  hipMemcpy(tempOrder, mesh->order->data_d, setSize * sizeof(int), hipMemcpyDeviceToHost);
  int *data_ptr = (int *)malloc(setSize * sizeof(int));
  hipMemcpy(data_ptr, glb_ind->data_d, setSize * sizeof(int), hipMemcpyDeviceToHost);

  int ind = global_ind;
  for(int i = 0; i < mesh->cells->size; i++) {
    int Np, Nfp;
    DGUtils::basic_constants(tempOrder[i], &Np, &Nfp);
    data_ptr[i] = ind;
    ind += Np;
  }

  hipMemcpy(glb_ind->data_d, data_ptr, setSize * sizeof(int), hipMemcpyHostToDevice);

  op_mpi_set_dirtybit_cuda(2, args);
  free(data_ptr);
  free(tempOrder);
}

void PoissonSolveHYPRE::setGlbInd() {
  int global_ind = 0;
  #ifdef INS_MPI
  global_ind = get_global_mat_start_ind(unknowns);
  #endif
  op_arg args[] = {
    op_arg_dat(mesh->order, -1, OP_ID, 1, "int", OP_READ),
    op_arg_dat(glb_ind, -1, OP_ID, 1, "int", OP_WRITE)
  };
  op_mpi_halo_exchanges_cuda(mesh->cells, 2, args);

  const int setSize = mesh->cells->size;
  int *tempOrder = (int *)malloc(setSize * sizeof(int));
  hipMemcpy(tempOrder, mesh->order->data_d, setSize * sizeof(int), hipMemcpyDeviceToHost);
  int *data_ptr = (int *)malloc(setSize * sizeof(int));
  hipMemcpy(data_ptr, glb_ind->data_d, setSize * sizeof(int), hipMemcpyDeviceToHost);

  int ind = global_ind;
  for(int i = 0; i < mesh->cells->size; i++) {
    int Np, Nfp;
    DGUtils::basic_constants(tempOrder[i], &Np, &Nfp);
    data_ptr[i] = ind;
    ind += Np;
  }

  hipMemcpy(glb_ind->data_d, data_ptr, setSize * sizeof(int), hipMemcpyHostToDevice);

  op_mpi_set_dirtybit_cuda(2, args);
  free(data_ptr);
  free(tempOrder);
}

void PoissonSolve::setMatrix() {
  if(pMatInit)
    MatDestroy(&pMat);

  MatCreate(PETSC_COMM_WORLD, &pMat);
  pMatInit = true;
  MatSetSizes(pMat, unknowns, unknowns, PETSC_DECIDE, PETSC_DECIDE);

  #ifdef INS_MPI
  MatSetType(pMat, MATMPIAIJCUSPARSE);
  MatMPIAIJSetPreallocation(pMat, DG_NP * 4, NULL, 0, NULL);
  #else
  MatSetType(pMat, MATSEQAIJCUSPARSE);
  MatSeqAIJSetPreallocation(pMat, DG_NP * 4, NULL);
  #endif
  MatSetOption(pMat, MAT_NEW_NONZERO_ALLOCATION_ERR, PETSC_FALSE);

  // Add cubature OP to Poisson matrix
  op_arg args[] = {
    op_arg_dat(pMatrix->op1, -1, OP_ID, DG_NP * DG_NP, "double", OP_READ),
    op_arg_dat(glb_ind, -1, OP_ID, 1, "int", OP_READ),
    op_arg_dat(mesh->order, -1, OP_ID, 1, "int", OP_READ)
  };
  op_mpi_halo_exchanges_cuda(mesh->cells, 3, args);

  const int setSize = mesh->cells->size;
  double *op1_data = (double *)malloc(DG_NP * DG_NP * setSize * sizeof(double));
  int *glb   = (int *)malloc(setSize * sizeof(int));
  int *order = (int *)malloc(setSize * sizeof(int));
  hipMemcpy(op1_data, pMatrix->op1->data_d, setSize * DG_NP * DG_NP * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(glb, glb_ind->data_d, setSize * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(order, mesh->order->data_d, setSize * sizeof(int), hipMemcpyDeviceToHost);
  op_mpi_set_dirtybit_cuda(3, args);

  MatSetOption(pMat, MAT_ROW_ORIENTED, PETSC_FALSE);

  for(int i = 0; i < setSize; i++) {
    int Np, Nfp;
    DGUtils::basic_constants(order[i], &Np, &Nfp);
    int currentRow = glb[i];
    int currentCol = glb[i];
    int idxm[DG_NP], idxn[DG_NP];
    for(int n = 0; n < DG_NP; n++) {
      idxm[n] = currentRow + n;
      idxn[n] = currentCol + n;
    }

    MatSetValues(pMat, Np, idxm, Np, idxn, &op1_data[i * DG_NP * DG_NP], INSERT_VALUES);
  }

  free(op1_data);
  free(glb);
  free(order);

  op_arg edge_args[] = {
    op_arg_dat(pMatrix->op2[0], -1, OP_ID, DG_NP * DG_NP, "double", OP_READ),
    op_arg_dat(pMatrix->op2[1], -1, OP_ID, DG_NP * DG_NP, "double", OP_READ),
    op_arg_dat(glb_indL, -1, OP_ID, 1, "int", OP_READ),
    op_arg_dat(glb_indR, -1, OP_ID, 1, "int", OP_READ),
    op_arg_dat(orderL, -1, OP_ID, 1, "int", OP_READ),
    op_arg_dat(orderR, -1, OP_ID, 1, "int", OP_READ)
  };
  op_mpi_halo_exchanges_cuda(mesh->edges, 6, edge_args);
  double *op2L_data = (double *)malloc(DG_NP * DG_NP * mesh->edges->size * sizeof(double));
  double *op2R_data = (double *)malloc(DG_NP * DG_NP * mesh->edges->size * sizeof(double));
  int *glb_l = (int *)malloc(mesh->edges->size * sizeof(int));
  int *glb_r = (int *)malloc(mesh->edges->size * sizeof(int));
  int *order_l = (int *)malloc(mesh->edges->size * sizeof(int));
  int *order_r = (int *)malloc(mesh->edges->size * sizeof(int));

  hipMemcpy(op2L_data, pMatrix->op2[0]->data_d, DG_NP * DG_NP * mesh->edges->size * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(op2R_data, pMatrix->op2[1]->data_d, DG_NP * DG_NP * mesh->edges->size * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(glb_l, glb_indL->data_d, mesh->edges->size * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(glb_r, glb_indR->data_d, mesh->edges->size * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(order_l, orderL->data_d, mesh->edges->size * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(order_r, orderR->data_d, mesh->edges->size * sizeof(int), hipMemcpyDeviceToHost);

  // Add Gauss OP and OPf to Poisson matrix
  for(int i = 0; i < mesh->edges->size; i++) {
    int leftRow = glb_l[i];
    int rightRow = glb_r[i];
    int NpL, NpR, Nfp;
    DGUtils::basic_constants(order_l[i], &NpL, &Nfp);
    DGUtils::basic_constants(order_r[i], &NpR, &Nfp);

    int idxl[DG_NP], idxr[DG_NP];
    for(int n = 0; n < DG_NP; n++) {
      idxl[n] = leftRow + n;
      idxr[n] = rightRow + n;
    }

    MatSetValues(pMat, NpL, idxl, NpR, idxr, &op2L_data[i * DG_NP * DG_NP], INSERT_VALUES);
    MatSetValues(pMat, NpR, idxr, NpL, idxl, &op2R_data[i * DG_NP * DG_NP], INSERT_VALUES);
  }

  free(op2L_data);
  free(op2R_data);
  free(glb_l);
  free(glb_r);
  free(order_l);
  free(order_r);

  op_mpi_set_dirtybit_cuda(6, edge_args);

  MatAssemblyBegin(pMat, MAT_FINAL_ASSEMBLY);
  MatAssemblyEnd(pMat, MAT_FINAL_ASSEMBLY);

}

void PoissonSolveHYPRE::setMatrix() {
  // Add cubature OP to Poisson matrix
  op_arg args[] = {
    op_arg_dat(pMatrix->op1, -1, OP_ID, DG_NP * DG_NP, "double", OP_READ),
    op_arg_dat(glb_ind, -1, OP_ID, 1, "int", OP_READ),
    op_arg_dat(mesh->order, -1, OP_ID, 1, "int", OP_READ)
  };
  op_mpi_halo_exchanges_cuda(mesh->cells, 3, args);

  const int setSize = mesh->cells->size;
  int *glb   = (int *)malloc(setSize * sizeof(int));
  int *order = (int *)malloc(setSize * sizeof(int));
  hipMemcpy(glb, glb_ind->data_d, setSize * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(order, mesh->order->data_d, setSize * sizeof(int), hipMemcpyDeviceToHost);

  int *idxm, *idxn, *nCols;
  hipMallocManaged(&idxm, DG_NP * setSize * sizeof(int));
  hipMallocManaged(&idxn, DG_NP * DG_NP * setSize * sizeof(int));
  hipMallocManaged(&nCols, DG_NP * setSize * sizeof(int));

  for(int i = 0; i < setSize; i++) {
    int currentRow = glb[i];
    int currentCol = glb[i];
    for(int n = 0; n < DG_NP; n++) {
      idxm[i * DG_NP + n] = currentRow + n;
      nCols[i * DG_NP + n] = DG_NP;
      for(int m = 0; m < DG_NP; m++) {
        idxn[i * DG_NP * DG_NP + n * DG_NP + m] = currentCol + m;
      }
    }
  }

  HYPRE_IJMatrixAddToValues(mat, DG_NP * setSize, nCols, idxm, idxn, (double *)pMatrix->op1->data_d);

  op_mpi_set_dirtybit_cuda(3, args);

  hipFree(idxm);
  hipFree(idxn);
  hipFree(nCols);

  free(glb);
  free(order);

  op_arg edge_args[] = {
    op_arg_dat(pMatrix->op2[0], -1, OP_ID, DG_NP * DG_NP, "double", OP_READ),
    op_arg_dat(pMatrix->op2[1], -1, OP_ID, DG_NP * DG_NP, "double", OP_READ),
    op_arg_dat(glb_indL, -1, OP_ID, 1, "int", OP_READ),
    op_arg_dat(glb_indR, -1, OP_ID, 1, "int", OP_READ),
    op_arg_dat(orderL, -1, OP_ID, 1, "int", OP_READ),
    op_arg_dat(orderR, -1, OP_ID, 1, "int", OP_READ)
  };
  op_mpi_halo_exchanges_cuda(mesh->edges, 6, edge_args);
  int *glb_l = (int *)malloc(mesh->edges->size * sizeof(int));
  int *glb_r = (int *)malloc(mesh->edges->size * sizeof(int));
  int *order_l = (int *)malloc(mesh->edges->size * sizeof(int));
  int *order_r = (int *)malloc(mesh->edges->size * sizeof(int));

  hipMemcpy(glb_l, glb_indL->data_d, mesh->edges->size * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(glb_r, glb_indR->data_d, mesh->edges->size * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(order_l, orderL->data_d, mesh->edges->size * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(order_r, orderR->data_d, mesh->edges->size * sizeof(int), hipMemcpyDeviceToHost);

  int *idxlm, *idxln, *idxrm, *idxrn, *nCols2;
  hipMallocManaged(&idxlm, DG_NP * mesh->edges->size * sizeof(int));
  hipMallocManaged(&idxln, DG_NP * DG_NP * mesh->edges->size * sizeof(int));
  hipMallocManaged(&idxrm, DG_NP * mesh->edges->size * sizeof(int));
  hipMallocManaged(&idxrn, DG_NP * DG_NP * mesh->edges->size * sizeof(int));
  hipMallocManaged(&nCols2, DG_NP * mesh->edges->size * sizeof(int));

  // Add Gauss OP and OPf to Poisson matrix
  for(int i = 0; i < mesh->edges->size; i++) {
    int leftRow = glb_l[i];
    int rightRow = glb_r[i];

    for(int n = 0; n < DG_NP; n++) {
      idxlm[i * DG_NP + n] = leftRow + n;
      idxrm[i * DG_NP + n] = rightRow + n;
      nCols2[i * DG_NP + n] = DG_NP;
      for(int m = 0; m < DG_NP; m++) {
        idxln[i * DG_NP * DG_NP + n * DG_NP + m] = rightRow + m;
        idxrn[i * DG_NP * DG_NP + n * DG_NP + m] = leftRow + m;
      }
    }
  }

  HYPRE_IJMatrixAddToValues(mat, DG_NP * mesh->edges->size, nCols2, idxlm, idxln, (double *)pMatrix->op2[0]->data_d);
  HYPRE_IJMatrixAddToValues(mat, DG_NP * mesh->edges->size, nCols2, idxrm, idxrn, (double *)pMatrix->op2[1]->data_d);

  hipFree(idxlm);
  hipFree(idxln);
  hipFree(idxrm);
  hipFree(idxrn);
  hipFree(nCols2);

  free(glb_l);
  free(glb_r);
  free(order_l);
  free(order_r);

  op_mpi_set_dirtybit_cuda(6, edge_args);
}
