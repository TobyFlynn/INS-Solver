#include "poisson.h"

#ifdef INS_MPI
#include "mpi_helper_func.h"
#endif

#include "dg_utils.h"

// Copy PETSc vec array to OP2 dat
void PoissonSolve::copy_vec_to_dat(op_dat dat, const double *dat_d) {
  op_arg copy_args[] = {
    op_arg_dat(dat, -1, OP_ID, DG_NP, "double", OP_WRITE),
    op_arg_dat(mesh->order, -1, OP_ID, 1, "int", OP_READ)
  };
  op_mpi_halo_exchanges_cuda(dat->set, 2, copy_args);

  int setSize = dat->set->size;
  int *tempOrder = (int *)malloc(setSize * sizeof(int));
  hipMemcpy(tempOrder, mesh->order->data_d, setSize * sizeof(int), hipMemcpyDeviceToHost);

  int vec_ind = 0;
  int block_start = 0;
  int block_count = 0;
  for(int i = 0; i < setSize; i++) {
    const int N = tempOrder[i];

    if(N == DG_ORDER) {
      if(block_count == 0) {
        block_start = i;
        block_count++;
        continue;
      } else {
        block_count++;
        continue;
      }
    } else {
      if(block_count != 0) {
        double *block_start_dat_c = (double *)dat->data_d + block_start * dat->dim;
        hipMemcpy(block_start_dat_c, dat_d + vec_ind, block_count * DG_NP * sizeof(double), hipMemcpyDeviceToDevice);
        vec_ind += DG_NP * block_count;
      }
      block_count = 0;
    }

    double *v_c = (double *)dat->data_d + i * dat->dim;
    int Np, Nfp;
    DGUtils::basic_constants(N, &Np, &Nfp);

    hipMemcpy(v_c, dat_d + vec_ind, Np * sizeof(double), hipMemcpyDeviceToDevice);
    vec_ind += Np;
  }

  if(block_count != 0) {
    double *block_start_dat_c = (double *)dat->data_d + block_start * dat->dim;
    hipMemcpy(block_start_dat_c, dat_d + vec_ind, block_count * DG_NP * sizeof(double), hipMemcpyDeviceToDevice);
    vec_ind += DG_NP * block_count;
  }

  op_mpi_set_dirtybit_cuda(2, copy_args);
}

// Copy OP2 dat to PETSc vec array
void PoissonSolve::copy_dat_to_vec(op_dat dat, double *dat_d) {
  op_arg copy_args[] = {
    op_arg_dat(dat, -1, OP_ID, DG_NP, "double", OP_READ),
    op_arg_dat(mesh->order, -1, OP_ID, 1, "int", OP_READ)
  };
  op_mpi_halo_exchanges_cuda(dat->set, 2, copy_args);

  int setSize = dat->set->size;
  int *tempOrder = (int *)malloc(setSize * sizeof(int));
  hipMemcpy(tempOrder, mesh->order->data_d, setSize * sizeof(int), hipMemcpyDeviceToHost);

  int vec_ind = 0;
  int block_start = 0;
  int block_count = 0;
  for(int i = 0; i < setSize; i++) {
    const int N = tempOrder[i];

    if(N == DG_ORDER) {
      if(block_count == 0) {
        block_start = i;
        block_count++;
        continue;
      } else {
        block_count++;
        continue;
      }
    } else {
      if(block_count != 0) {
        const double *block_start_dat_c = (double *)dat->data_d + block_start * dat->dim;
        hipMemcpy(dat_d + vec_ind, block_start_dat_c, block_count * DG_NP * sizeof(double), hipMemcpyDeviceToDevice);
        vec_ind += DG_NP * block_count;
      }
      block_count = 0;
    }

    const double *v_c = (double *)dat->data_d + i * dat->dim;
    int Np, Nfp;
    DGUtils::basic_constants(N, &Np, &Nfp);

    hipMemcpy(dat_d + vec_ind, v_c, Np * sizeof(double), hipMemcpyDeviceToDevice);
    vec_ind += Np;
  }

  if(block_count != 0) {
    const double *block_start_dat_c = (double *)dat->data_d + block_start * dat->dim;
    hipMemcpy(dat_d + vec_ind, block_start_dat_c, block_count * DG_NP * sizeof(double), hipMemcpyDeviceToDevice);
    vec_ind += DG_NP * block_count;
  }

  op_mpi_set_dirtybit_cuda(2, copy_args);
}

// Create a PETSc vector for GPUs
void PoissonSolve::create_vec(Vec *v) {
  VecCreate(PETSC_COMM_WORLD, v);
  VecSetType(*v, VECCUDA);
  VecSetSizes(*v, unknowns, PETSC_DECIDE);
}

// Destroy a PETSc vector
void PoissonSolve::destroy_vec(Vec *v) {
  VecDestroy(v);
}

// Load a PETSc vector with values from an OP2 dat for GPUs
void PoissonSolve::load_vec(Vec *v, op_dat v_dat) {
  double *v_ptr;
  VecCUDAGetArray(*v, &v_ptr);

  copy_dat_to_vec(v_dat, v_ptr);

  VecCUDARestoreArray(*v, &v_ptr);
}

// Load an OP2 dat with the values from a PETSc vector for GPUs
void PoissonSolve::store_vec(Vec *v, op_dat v_dat) {
  const double *v_ptr;
  VecCUDAGetArrayRead(*v, &v_ptr);

  copy_vec_to_dat(v_dat, v_ptr);

  VecCUDARestoreArrayRead(*v, &v_ptr);
}

PetscErrorCode matAMult(Mat A, Vec x, Vec y) {
  PoissonSolve *poisson;
  MatShellGetContext(A, &poisson);
  const double *x_ptr;
  double *y_ptr;
  VecCUDAGetArrayRead(x, &x_ptr);
  VecCUDAGetArray(y, &y_ptr);

  poisson->calc_rhs(x_ptr, y_ptr);

  VecCUDARestoreArrayRead(x, &x_ptr);
  VecCUDARestoreArray(y, &y_ptr);
  return 0;
}

void PoissonSolve::create_shell_mat(Mat *m) {
  MatCreateShell(PETSC_COMM_WORLD, unknowns, unknowns, PETSC_DETERMINE, PETSC_DETERMINE, this, m);
  MatShellSetOperation(*m, MATOP_MULT, (void(*)(void))matAMult);
  MatShellSetVecType(*m, VECCUDA);
}

PetscErrorCode precon(PC pc, Vec x, Vec y) {
  PoissonSolve *poisson;
  PCShellGetContext(pc, (void **)&poisson);
  const double *x_ptr;
  double *y_ptr;
  VecCUDAGetArrayRead(x, &x_ptr);
  VecCUDAGetArray(y, &y_ptr);

  poisson->precond(x_ptr, y_ptr);

  VecCUDARestoreArrayRead(x, &x_ptr);
  VecCUDARestoreArray(y, &y_ptr);
  return 0;
}

void PoissonSolve::set_shell_pc(PC pc) {
  PCShellSetApply(pc, precon);
  PCShellSetContext(pc, this);
}

void PoissonSolve::setGlbInd() {
  int global_ind = 0;
  #ifdef INS_MPI
  global_ind = get_global_start_index(glb_ind->set);
  #endif
  op_arg args[] = {
    op_arg_dat(glb_ind, -1, OP_ID, 1, "int", OP_WRITE)
  };
  op_mpi_halo_exchanges_cuda(mesh->cells, 1, args);
  int *data_ptr = (int *)malloc(mesh->cells->size * sizeof(int));
  hipMemcpy(data_ptr, glb_ind->data_d, glb_ind->set->size * sizeof(int), hipMemcpyDeviceToHost);
  for(int i = 0; i < mesh->cells->size; i++) {
    data_ptr[i] = global_ind + i;
  }
  hipMemcpy(glb_ind->data_d, data_ptr, glb_ind->set->size * sizeof(int), hipMemcpyHostToDevice);
  op_mpi_set_dirtybit_cuda(1, args);
  free(data_ptr);
}

void PoissonSolve::setMatrix() {
  if(pMatInit) {
    MatDestroy(&pMat);
  }
  MatCreate(PETSC_COMM_WORLD, &pMat);
  pMatInit = true;
  MatSetSizes(pMat, unknowns, unknowns, PETSC_DECIDE, PETSC_DECIDE);

  #ifdef INS_MPI
  MatSetType(pMat, MATMPIAIJCUSPARSE);
  MatMPIAIJSetPreallocation(pMat, DG_NP * 4, NULL, 0, NULL);
  #else
  MatSetType(pMat, MATSEQAIJCUSPARSE);
  MatSeqAIJSetPreallocation(pMat, DG_NP * 4, NULL);
  #endif
  MatSetOption(pMat, MAT_NEW_NONZERO_ALLOCATION_ERR, PETSC_FALSE);

  // Add cubature OP to Poisson matrix
  op_arg args[] = {
    op_arg_dat(op1, -1, OP_ID, DG_NP * DG_NP, "double", OP_READ),
    op_arg_dat(glb_ind, -1, OP_ID, 1, "int", OP_READ),
    op_arg_dat(mesh->order, -1, OP_ID, 1, "int", OP_READ)
  };
  op_mpi_halo_exchanges_cuda(mesh->cells, 3, args);
  double *op1_data = (double *)malloc(DG_NP * DG_NP * mesh->cells->size * sizeof(double));
  int *glb   = (int *)malloc(mesh->cells->size * sizeof(int));
  int *order = (int *)malloc(mesh->cells->size * sizeof(int));
  hipMemcpy(op1_data, op1->data_d, op1->set->size * DG_NP * DG_NP * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(glb, glb_ind->data_d, glb_ind->set->size * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(order, mesh->order->data_d, mesh->order->set->size * sizeof(int), hipMemcpyDeviceToHost);
  op_mpi_set_dirtybit_cuda(3, args);

  int currentRow = 0;
  int currentCol = 0;
  for(int i = 0; i < mesh->cells->size; i++) {
    int global_ind = glb[i];
    int Np, Nfp;
    DGUtils::basic_constants(order[i], &Np, &Nfp);

    // Convert data to row major format
    for(int m = 0; m < Np; m++) {
      for(int n = 0; n < Np; n++) {
        int row = currentRow + m;
        int col = currentCol + n;
        double val = op1_data[i * DG_NP * DG_NP + m + n * Np];
        MatSetValues(pMat, 1, &row, 1, &col, &val, INSERT_VALUES);
      }
    }

    glb[i] = currentRow;
    currentRow += Np;
    currentCol += Np;
  }

  free(op1_data);
  // free(glb);
  // free(order);

  op_arg edge_args[] = {
    op_arg_dat(op2[0], -1, OP_ID, DG_NP * DG_NP, "double", OP_READ),
    op_arg_dat(op2[1], -1, OP_ID, DG_NP * DG_NP, "double", OP_READ),
    op_arg_dat(glb_indL, -1, OP_ID, 1, "int", OP_READ),
    op_arg_dat(glb_indR, -1, OP_ID, 1, "int", OP_READ)
  };
  op_mpi_halo_exchanges_cuda(mesh->edges, 4, edge_args);
  double *op2L_data = (double *)malloc(DG_NP * DG_NP * mesh->edges->size * sizeof(double));
  double *op2R_data = (double *)malloc(DG_NP * DG_NP * mesh->edges->size * sizeof(double));
  int *glb_l = (int *)malloc(mesh->edges->size * sizeof(int));
  int *glb_r = (int *)malloc(mesh->edges->size * sizeof(int));

  hipMemcpy(op2L_data, op2[0]->data_d, DG_NP * DG_NP * mesh->edges->size * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(op2R_data, op2[1]->data_d, DG_NP * DG_NP * mesh->edges->size * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(glb_l, glb_indL->data_d, mesh->edges->size * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(glb_r, glb_indR->data_d, mesh->edges->size * sizeof(int), hipMemcpyDeviceToHost);

  // Add Gauss OP and OPf to Poisson matrix
  for(int i = 0; i < mesh->edges->size; i++) {
    int leftElement = glb_l[i];
    int rightElement = glb_r[i];
    int leftRow  = glb[leftElement];
    int rightRow = glb[rightElement];
    int NpL, NpR, Nfp;
    DGUtils::basic_constants(order[leftElement], &NpL, &Nfp);
    DGUtils::basic_constants(order[rightElement], &NpR, &Nfp);

    // Gauss OPf
    // Convert data to row major format
    for(int m = 0; m < NpL; m++) {
      for(int n = 0; n < NpR; n++) {
        int row = leftRow + m;
        int col = rightRow + n;
        double val = op2L_data[i * DG_NP * DG_NP + m + n * NpL];
        MatSetValues(pMat, 1, &row, 1, &col, &val, INSERT_VALUES);
      }
    }
    // Convert data to row major format
    for(int m = 0; m < NpR; m++) {
      for(int n = 0; n < NpL; n++) {
        int row = rightRow + m;
        int col = leftRow + n;
        double val = op2R_data[i * DG_NP * DG_NP + m + n * NpR];
        MatSetValues(pMat, 1, &row, 1, &col, &val, INSERT_VALUES);
      }
    }
  }

  free(op2L_data);
  free(op2R_data);
  free(glb_l);
  free(glb_r);

  free(glb);
  free(order);

  op_mpi_set_dirtybit_cuda(4, edge_args);

  MatAssemblyBegin(pMat, MAT_FINAL_ASSEMBLY);
  MatAssemblyEnd(pMat, MAT_FINAL_ASSEMBLY);

}
