#include "constants.h"

Constants::Constants() {
  // Cubature constants
  hipMalloc((void**)&cubDr_d, 46 * 15 * sizeof(double));
  hipMemcpy(cubDr_d, cubDr_g, 46 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&cubDs_d, 46 * 15 * sizeof(double));
  hipMemcpy(cubDs_d, cubDs_g, 46 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&cubV_d, 46 * 15 * sizeof(double));
  hipMemcpy(cubV_d, cubV_g, 46 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&cubVDr_d, 46 * 15 * sizeof(double));
  hipMemcpy(cubVDr_d, cubVDr_g, 46 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&cubVDs_d, 46 * 15 * sizeof(double));
  hipMemcpy(cubVDs_d, cubVDs_g, 46 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&cubW_d, 46 * sizeof(double));
  hipMemcpy(cubW_d, cubW_d, 46 * sizeof(double), hipMemcpyHostToDevice);
  // Grad constants
  hipMalloc((void**)&Dr_d, 15 * 15 * sizeof(double));
  hipMemcpy(Dr_d, Dr_g, 15 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&Drw_d, 15 * 15 * sizeof(double));
  hipMemcpy(Drw_d, Drw_g, 15 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&Ds_d, 15 * 15 * sizeof(double));
  hipMemcpy(Ds_d, Ds_g, 15 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&Dsw_d, 15 * 15 * sizeof(double));
  hipMemcpy(Dsw_d, Dsw_g, 15 * 15 * sizeof(double), hipMemcpyHostToDevice);
  // Gauss constants
  hipMalloc((void**)&gaussW_d, 7 * sizeof(double));
  hipMemcpy(gaussW_d, gaussW_g, 7 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF0Dr_d, 7 * 15 * sizeof(double));
  hipMemcpy(gF0Dr_d, gF0Dr_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF0DrR_d, 7 * 15 * sizeof(double));
  hipMemcpy(gF0DrR_d, gF0DrR_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF0Ds_d, 7 * 15 * sizeof(double));
  hipMemcpy(gF0Ds_d, gF0Ds_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF0DsR_d, 7 * 15 * sizeof(double));
  hipMemcpy(gF0DsR_d, gF0DsR_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF1Dr_d, 7 * 15 * sizeof(double));
  hipMemcpy(gF1Dr_d, gF1Dr_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF1DrR_d, 7 * 15 * sizeof(double));
  hipMemcpy(gF1DrR_d, gF1DrR_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF1Ds_d, 7 * 15 * sizeof(double));
  hipMemcpy(gF1Ds_d, gF1Ds_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF1DsR_d, 7 * 15 * sizeof(double));
  hipMemcpy(gF1DsR_d, gF1DsR_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF2Dr_d, 7 * 15 * sizeof(double));
  hipMemcpy(gF2Dr_d, gF2Dr_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF2DrR_d, 7 * 15 * sizeof(double));
  hipMemcpy(gF2DrR_d, gF2DrR_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF2Ds_d, 7 * 15 * sizeof(double));
  hipMemcpy(gF2Ds_d, gF2Ds_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF2DsR_d, 7 * 15 * sizeof(double));
  hipMemcpy(gF2DsR_d, gF2DsR_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gFInterp0_d, 7 * 15 * sizeof(double));
  hipMemcpy(gFInterp0_d, gFInterp0_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gFInterp0R_d, 7 * 15 * sizeof(double));
  hipMemcpy(gFInterp0R_d, gFInterp0R_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gFInterp1_d, 7 * 15 * sizeof(double));
  hipMemcpy(gFInterp1_d, gFInterp1_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gFInterp1R_d, 7 * 15 * sizeof(double));
  hipMemcpy(gFInterp1R_d, gFInterp1R_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gFInterp2_d, 7 * 15 * sizeof(double));
  hipMemcpy(gFInterp2_d, gFInterp2_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gFInterp2R_d, 7 * 15 * sizeof(double));
  hipMemcpy(gFInterp2R_d, gFInterp2R_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gInterp_d, 21 * 15 * sizeof(double));
  hipMemcpy(gInterp_d, gInterp_g, 21 * 15 * sizeof(double), hipMemcpyHostToDevice);
  // Other constants
  hipMalloc((void**)&invMass_d, 15 * 15 * sizeof(double));
  hipMemcpy(invMass_d, invMass_g, 15 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&LIFT_d, 15 * 15 * sizeof(double));
  hipMemcpy(LIFT_d, LIFT_g, 15 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&MASS_d, 15 * 15 * sizeof(double));
  hipMemcpy(MASS_d, MASS_g, 15 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&r_d, 15 * sizeof(double));
  hipMemcpy(r_d, r_g, 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&s_d, 15 * sizeof(double));
  hipMemcpy(s_d, s_g, 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&ones_d, 15 * sizeof(double));
  hipMemcpy(ones_d, ones_g, 15 * sizeof(double), hipMemcpyHostToDevice);

  hipblasCreate(&handle);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
}

Constants::~Constants() {
  // Cubature constants
  hipFree(cubDr_d);
  hipFree(cubDs_d);
  hipFree(cubV_d);
  hipFree(cubVDr_d);
  hipFree(cubVDs_d);
  hipFree(cubW_d);
  // Grad constants
  hipFree(Dr_d);
  hipFree(Drw_d);
  hipFree(Ds_d);
  hipFree(Dsw_d);
  // Gauss constants
  hipFree(gaussW_d);
  hipFree(gF0Dr_d);
  hipFree(gF0DrR_d);
  hipFree(gF0Ds_d);
  hipFree(gF0DsR_d);
  hipFree(gF1Dr_d);
  hipFree(gF1DrR_d);
  hipFree(gF1Ds_d);
  hipFree(gF1DsR_d);
  hipFree(gF2Dr_d);
  hipFree(gF2DrR_d);
  hipFree(gF2Ds_d);
  hipFree(gF2DsR_d);
  hipFree(gFInterp0_d);
  hipFree(gFInterp0R_d);
  hipFree(gFInterp1_d);
  hipFree(gFInterp1R_d);
  hipFree(gFInterp2_d);
  hipFree(gFInterp2R_d);
  hipFree(gInterp_d);
  // Other constants
  hipFree(invMass_d);
  hipFree(LIFT_d);
  hipFree(MASS_d);
  hipFree(r_d);
  hipFree(s_d);
  hipFree(ones_d);

  hipblasDestroy(handle);
}
