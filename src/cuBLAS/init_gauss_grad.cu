#include "hipblas.h"

#include "op_seq.h"
#include "../blas_calls.h"

inline void cublas_init_gauss_grad(hipblasHandle_t handle, const int numCells,
                        const double *x_d, const double *y_d, double *gxr_d,
                        double *gxs_d, double *gyr_d, double *gys_d) {
  double *g0Dr_d;
  hipMalloc((void**)&g0Dr_d, 7 * 15 * sizeof(double));
  hipMemcpy(g0Dr_d, gF0Dr, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  double *g0Ds_d;
  hipMalloc((void**)&g0Ds_d, 7 * 15 * sizeof(double));
  hipMemcpy(g0Ds_d, gF0Ds, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  double *g1Dr_d;
  hipMalloc((void**)&g1Dr_d, 7 * 15 * sizeof(double));
  hipMemcpy(g1Dr_d, gF1Dr, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  double *g1Ds_d;
  hipMalloc((void**)&g1Ds_d, 7 * 15 * sizeof(double));
  hipMemcpy(g1Ds_d, gF1Ds, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  double *g2Dr_d;
  hipMalloc((void**)&g2Dr_d, 7 * 15 * sizeof(double));
  hipMemcpy(g2Dr_d, gF2Dr, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  double *g2Ds_d;
  hipMalloc((void**)&g2Ds_d, 7 * 15 * sizeof(double));
  hipMemcpy(g2Ds_d, gF2Ds, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);

  // Calc Grad Matrices
  double alpha = 1.0;
  double beta = 0.0;
  for(int c = 0; c < numCells; c++) {
    const double *x = x_d + c * 15;
    const double *y = y_d + c * 15;
    double *gxr = gxr_d + c * 21;
    double *gxs = gxs_d + c * 21;
    double *gyr = gyr_d + c * 21;
    double *gys = gys_d + c * 21;

    // Face 0
    hipblasDgemv(handle, HIPBLAS_OP_T, 15, 7, &alpha, g0Dr_d, 15, x, 1, &beta, gxr, 1);
    hipblasDgemv(handle, HIPBLAS_OP_T, 15, 7, &alpha, g0Ds_d, 15, x, 1, &beta, gxs, 1);
    hipblasDgemv(handle, HIPBLAS_OP_T, 15, 7, &alpha, g0Dr_d, 15, y, 1, &beta, gyr, 1);
    hipblasDgemv(handle, HIPBLAS_OP_T, 15, 7, &alpha, g0Ds_d, 15, y, 1, &beta, gys, 1);

    // Face 1
    hipblasDgemv(handle, HIPBLAS_OP_T, 15, 7, &alpha, g1Dr_d, 15, x, 1, &beta, gxr + 7, 1);
    hipblasDgemv(handle, HIPBLAS_OP_T, 15, 7, &alpha, g1Ds_d, 15, x, 1, &beta, gxs + 7, 1);
    hipblasDgemv(handle, HIPBLAS_OP_T, 15, 7, &alpha, g1Dr_d, 15, y, 1, &beta, gyr + 7, 1);
    hipblasDgemv(handle, HIPBLAS_OP_T, 15, 7, &alpha, g1Ds_d, 15, y, 1, &beta, gys + 7, 1);

    // Face 2
    hipblasDgemv(handle, HIPBLAS_OP_T, 15, 7, &alpha, g2Dr_d, 15, x, 1, &beta, gxr + 14, 1);
    hipblasDgemv(handle, HIPBLAS_OP_T, 15, 7, &alpha, g2Ds_d, 15, x, 1, &beta, gxs + 14, 1);
    hipblasDgemv(handle, HIPBLAS_OP_T, 15, 7, &alpha, g2Dr_d, 15, y, 1, &beta, gyr + 14, 1);
    hipblasDgemv(handle, HIPBLAS_OP_T, 15, 7, &alpha, g2Ds_d, 15, y, 1, &beta, gys + 14, 1);
  }

  hipFree(g0Dr_d);
  hipFree(g0Ds_d);
  hipFree(g1Dr_d);
  hipFree(g1Ds_d);
  hipFree(g2Dr_d);
  hipFree(g2Ds_d);
}

void init_gauss_grad_blas(INSData *nsData, GaussData *gaussData) {
  // Initialise cuBLAS
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
  // Make sure OP2 data is in the right place
  op_arg init_grad_args[] = {
    op_arg_dat(nsData->x, -1, OP_ID, 15, "double", OP_READ),
    op_arg_dat(nsData->y, -1, OP_ID, 15, "double", OP_READ),
    op_arg_dat(gaussData->rx, -1, OP_ID, 21, "double", OP_WRITE),
    op_arg_dat(gaussData->sx, -1, OP_ID, 21, "double", OP_WRITE),
    op_arg_dat(gaussData->ry, -1, OP_ID, 21, "double", OP_WRITE),
    op_arg_dat(gaussData->sy, -1, OP_ID, 21, "double", OP_WRITE)
  };
  op_mpi_halo_exchanges_cuda(nsData->cells, 6, init_grad_args);

  cublas_init_gauss_grad(handle, nsData->numCells, (double *)nsData->x->data_d,
                   (double *)nsData->y->data_d, (double *)gaussData->rx->data_d,
                   (double *)gaussData->sx->data_d, (double *)gaussData->ry->data_d,
                   (double *)gaussData->sy->data_d);

  // Set correct dirty bits for OP2
  op_mpi_set_dirtybit_cuda(6, init_grad_args);
  // Free resources used by cuBLAS
  hipblasDestroy(handle);
}
