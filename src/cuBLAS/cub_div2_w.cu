#include "hipblas.h"

#include "op_seq.h"
#include "../blas_calls.h"

inline void cublas_cub_div2_w(hipblasHandle_t handle, const int numCells,
                            const double *temp0, const double *temp1,
                            const double *temp2, const double *temp3,
                            double *res_d) {
  // HIPBLAS_OP_T because cublas is column major but constants are stored row major
  double alpha = 1.0;
  double beta = 0.0;
  double beta2 = 1.0;
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 15, numCells, 46, &alpha, constants->cubDr_d, 15, temp0, 46, &beta, res_d, 15);
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 15, numCells, 46, &alpha, constants->cubDs_d, 15, temp1, 46, &beta2, res_d, 15);

  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 15, numCells, 46, &alpha, constants->cubDr_d, 15, temp2, 46, &beta2, res_d, 15);
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 15, numCells, 46, &alpha, constants->cubDs_d, 15, temp3, 46, &beta2, res_d, 15);
}

void cub_div_w_blas2(INSData *data, CubatureData *cubatureData, op_dat res) {
  // Make sure OP2 data is in the right place
  op_arg div_args[] = {
    op_arg_dat(cubatureData->op_temps[0], -1, OP_ID, 46, "double", OP_READ),
    op_arg_dat(cubatureData->op_temps[1], -1, OP_ID, 46, "double", OP_READ),
    op_arg_dat(cubatureData->op_temps[2], -1, OP_ID, 46, "double", OP_READ),
    op_arg_dat(cubatureData->op_temps[3], -1, OP_ID, 46, "double", OP_READ),
    op_arg_dat(res, -1, OP_ID, 15, "double", OP_WRITE)
  };
  op_mpi_halo_exchanges_cuda(data->cells, 5, div_args);

  cublas_cub_div2_w(constants->handle, data->numCells, (double *)cubatureData->op_temps[0]->data_d,
                  (double *)cubatureData->op_temps[1]->data_d, (double *)cubatureData->op_temps[2]->data_d,
                  (double *)cubatureData->op_temps[3]->data_d, (double *)res->data_d);

  // Set correct dirty bits for OP2
  op_mpi_set_dirtybit_cuda(5, div_args);
}
