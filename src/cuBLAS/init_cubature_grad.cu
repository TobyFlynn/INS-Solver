#include "hipblas.h"

#include "op_seq.h"
#include "../blas_calls.h"

inline void cublas_init_cubature(hipblasHandle_t handle, const int numCells,
                        const double *x_d, const double *y_d, double *cxr_d,
                        double *cxs_d, double *cyr_d, double *cys_d) {
  double alpha = 1.0;
  double beta = 0.0;
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 46, numCells, 15, &alpha, constants->cubVDr_d, 15, x_d, 15, &beta, cxr_d, 46);
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 46, numCells, 15, &alpha, constants->cubVDs_d, 15, x_d, 15, &beta, cxs_d, 46);
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 46, numCells, 15, &alpha, constants->cubVDr_d, 15, y_d, 15, &beta, cyr_d, 46);
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 46, numCells, 15, &alpha, constants->cubVDs_d, 15, y_d, 15, &beta, cys_d, 46);
}

void init_cubature_grad_blas(INSData *nsData, CubatureData *cubData) {
  // Make sure OP2 data is in the right place
  op_arg init_cubature_args[] = {
    op_arg_dat(nsData->x, -1, OP_ID, 15, "double", OP_READ),
    op_arg_dat(nsData->y, -1, OP_ID, 15, "double", OP_READ),
    op_arg_dat(cubData->rx, -1, OP_ID, 46, "double", OP_WRITE),
    op_arg_dat(cubData->sx, -1, OP_ID, 46, "double", OP_WRITE),
    op_arg_dat(cubData->ry, -1, OP_ID, 46, "double", OP_WRITE),
    op_arg_dat(cubData->sy, -1, OP_ID, 46, "double", OP_WRITE)
  };
  op_mpi_halo_exchanges_cuda(nsData->cells, 6, init_cubature_args);

  cublas_init_cubature(constants->handle, nsData->numCells, (double *)nsData->x->data_d,
                   (double *)nsData->y->data_d, (double *)cubData->rx->data_d,
                   (double *)cubData->sx->data_d, (double *)cubData->ry->data_d,
                   (double *)cubData->sy->data_d);

  // Set correct dirty bits for OP2
  op_mpi_set_dirtybit_cuda(6, init_cubature_args);
}
