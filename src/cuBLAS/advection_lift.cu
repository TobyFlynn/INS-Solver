#include "hipblas.h"

#include "op_seq.h"
#include "../blas_calls.h"

inline void cublas_advection_lift(hipblasHandle_t handle, const int numCells,
                                  const double *flux0_d, const double *flux1_d,
                                  double *N0_d, double *N1_d) {
  // double *LIFT_d;
  // hipMalloc((void**)&LIFT_d, 15 * 15 * sizeof(double));
  // hipMemcpy(LIFT_d, LIFT, 15 * 15 * sizeof(double), hipMemcpyHostToDevice);

  // HIPBLAS_OP_T because cublas is column major but constants are stored row major
  double alpha = 1.0;
  double beta = 1.0;
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 15, numCells, 15, &alpha, constants->LIFT_d, 15, flux0_d, 15, &beta, N0_d, 15);
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 15, numCells, 15, &alpha, constants->LIFT_d, 15, flux1_d, 15, &beta, N1_d, 15);
  // hipFree(LIFT_d);
}

void advection_lift_blas(INSData *nsData, int ind) {
  // Initialise cuBLAS
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
  // Make sure OP2 data is in the right place
  op_arg advec_lift_args[] = {
    op_arg_dat(nsData->flux[0], -1, OP_ID, 15, "double", OP_READ),
    op_arg_dat(nsData->flux[1], -1, OP_ID, 15, "double", OP_READ),
    op_arg_dat(nsData->N[ind][0], -1, OP_ID, 15, "double", OP_RW),
    op_arg_dat(nsData->N[ind][1], -1, OP_ID, 15, "double", OP_RW)
  };
  op_mpi_halo_exchanges_cuda(nsData->cells, 4, advec_lift_args);

  cublas_advection_lift(handle, nsData->numCells, (double *)nsData->flux[0]->data_d,
                     (double *)nsData->flux[1]->data_d, (double *)nsData->N[ind][0]->data_d,
                     (double *)nsData->N[ind][1]->data_d);

  // Set correct dirty bits for OP2
  op_mpi_set_dirtybit_cuda(4, advec_lift_args);
  // Free resources used by cuBLAS
  hipblasDestroy(handle);
}
