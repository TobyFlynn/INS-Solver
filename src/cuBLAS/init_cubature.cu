#include "hipblas.h"

#include "op_seq.h"
#include "../blas_calls.h"

inline void cublas_init_cubature(hipblasHandle_t handle, const int numCells,
                        const double *x_d, const double *y_d, double *cxr_d,
                        double *cxs_d, double *cyr_d, double *cys_d) {
  double *cubDr_d;
  hipMalloc((void**)&cubDr_d, 46 * 15 * sizeof(double));
  hipMemcpy(cubDr_d, cubDr, 46 * 15 * sizeof(double), hipMemcpyHostToDevice);

  double *cubDs_d;
  hipMalloc((void**)&cubDs_d, 46 * 15 * sizeof(double));
  hipMemcpy(cubDs_d, cubDs, 46 * 15 * sizeof(double), hipMemcpyHostToDevice);

  double alpha = 1.0;
  double beta = 0.0;
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 46, numCells, 15, &alpha, cubDr_d, 15, x_d, 15, &beta, cxr_d, 46);
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 46, numCells, 15, &alpha, cubDs_d, 15, x_d, 15, &beta, cxs_d, 46);
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 46, numCells, 15, &alpha, cubDr_d, 15, y_d, 15, &beta, cyr_d, 46);
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 46, numCells, 15, &alpha, cubDs_d, 15, y_d, 15, &beta, cys_d, 46);

  hipFree(cubDr_d);
  hipFree(cubDs_d);
}

void init_cubature_blas(INSData *nsData, CubatureData *cubData) {
  // Initialise cuBLAS
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
  // Make sure OP2 data is in the right place
  op_arg init_cubature_args[] = {
    op_arg_dat(nsData->x, -1, OP_ID, 15, "double", OP_READ),
    op_arg_dat(nsData->y, -1, OP_ID, 15, "double", OP_READ),
    op_arg_dat(cubData->rx, -1, OP_ID, 46, "double", OP_WRITE),
    op_arg_dat(cubData->sx, -1, OP_ID, 46, "double", OP_WRITE),
    op_arg_dat(cubData->ry, -1, OP_ID, 46, "double", OP_WRITE),
    op_arg_dat(cubData->sy, -1, OP_ID, 46, "double", OP_WRITE)
  };
  op_mpi_halo_exchanges_cuda(nsData->cells, 6, init_cubature_args);

  cublas_init_cubature(handle, nsData->numCells, (double *)nsData->x->data_d,
                   (double *)nsData->y->data_d, (double *)cubData->rx->data_d,
                   (double *)cubData->sx->data_d, (double *)cubData->ry->data_d,
                   (double *)cubData->sy->data_d);

  // Set correct dirty bits for OP2
  op_mpi_set_dirtybit_cuda(6, init_cubature_args);
  // Free resources used by cuBLAS
  hipblasDestroy(handle);
}
