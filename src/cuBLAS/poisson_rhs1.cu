#include "hipblas.h"

#include "op_seq.h"
#include "../blas_calls.h"

inline void cublas_poisson_rhs1(hipblasHandle_t handle, const int numCells,
                                const double *fluxX_d, const double *fluxY_d,
                                double *dudx_d, double *dudy_d, double *qx_d,
                                double *qy_d, double *gradx_d, double *grady_d) {
  // HIPBLAS_OP_T because cublas is column major but constants are stored row major
  double alpha = -1.0;
  double alpha2 = 1.0;
  double beta = 1.0;
  double beta2 = 0.0;
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 15, numCells, 15, &alpha2, constants->invMass_d, 15, dudx_d, 15, &beta2, gradx_d, 15);
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 15, numCells, 15, &alpha2, constants->invMass_d, 15, dudy_d, 15, &beta2, grady_d, 15);

  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 15, numCells, 21, &alpha, constants->gInterp_d, 15, fluxX_d, 21, &beta, dudx_d, 15);
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 15, numCells, 21, &alpha, constants->gInterp_d, 15, fluxY_d, 21, &beta, dudy_d, 15);

  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 15, numCells, 15, &alpha2, constants->invMass_d, 15, dudx_d, 15, &beta2, qx_d, 15);
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 15, numCells, 15, &alpha2, constants->invMass_d, 15, dudy_d, 15, &beta2, qy_d, 15);
}

void poisson_rhs_blas1(INSData *data, Poisson_MF *poisson) {
  // Make sure OP2 data is in the right place
  op_arg poisson_args[] = {
    op_arg_dat(poisson->uFluxX, -1, OP_ID, 21, "double", OP_READ),
    op_arg_dat(poisson->uFluxY, -1, OP_ID, 21, "double", OP_READ),
    op_arg_dat(poisson->dudx, -1, OP_ID, 15, "double", OP_RW),
    op_arg_dat(poisson->dudy, -1, OP_ID, 15, "double", OP_RW),
    op_arg_dat(poisson->qx, -1, OP_ID, 15, "double", OP_WRITE),
    op_arg_dat(poisson->qy, -1, OP_ID, 15, "double", OP_WRITE),
    op_arg_dat(poisson->gradx, -1, OP_ID, 15, "double", OP_WRITE),
    op_arg_dat(poisson->grady, -1, OP_ID, 15, "double", OP_WRITE)
  };
  op_mpi_halo_exchanges_cuda(data->cells, 8, poisson_args);

  cublas_poisson_rhs1(constants->handle, data->numCells, (double *)poisson->uFluxX->data_d,
                      (double *)poisson->uFluxY->data_d, (double *)poisson->dudx->data_d,
                      (double *)poisson->dudy->data_d, (double *)poisson->qx->data_d,
                      (double *)poisson->qy->data_d, (double *)poisson->gradx->data_d,
                      (double *)poisson->grady->data_d);

  // Set correct dirty bits for OP2
  op_mpi_set_dirtybit_cuda(8, poisson_args);
}
