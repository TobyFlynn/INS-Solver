#include "hipblas.h"

#include "op_seq.h"
#include "../blas_calls.h"

inline void cublas_poisson_rhs1(hipblasHandle_t handle, const int numCells,
                                const double *fluxX_d, const double *fluxY_d,
                                double *qx_d, double *qy_d) {
  double *interp_d;
  hipMalloc((void**)&interp_d, 21 * 15 * sizeof(double));
  hipMemcpy(interp_d, gInterp, 21 * 15 * sizeof(double), hipMemcpyHostToDevice);

  double *invMass_d;
  hipMalloc((void**)&invMass_d, 15 * 15 * sizeof(double));
  hipMemcpy(invMass_d, invMass, 15 * 15 * sizeof(double), hipMemcpyHostToDevice);

  double *temp_d;
  hipMalloc((void**)&temp_d, 15 * numCells * sizeof(double));

  // HIPBLAS_OP_T because cublas is column major but constants are stored row major
  double alpha = 1.0;
  double beta = -1.0;
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 15, numCells, 21, &alpha, interp_d, 15, fluxX_d, 21, &beta, qx_d, 15);
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 15, numCells, 21, &alpha, interp_d, 15, fluxY_d, 21, &beta, qy_d, 15);

  double beta2 = 0.0;
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 15, numCells, 15, &alpha, invMass_d, 15, qx_d, 15, &beta2, temp_d, 15);
  hipMemcpy(qx_d, temp_d, 15 * numCells * sizeof(double), hipMemcpyDeviceToDevice);

  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 15, numCells, 15, &alpha, invMass_d, 15, qy_d, 15, &beta2, temp_d, 15);
  hipMemcpy(qy_d, temp_d, 15 * numCells * sizeof(double), hipMemcpyDeviceToDevice);

  hipFree(interp_d);
  hipFree(invMass_d);
  hipFree(temp_d);
}

void poisson_rhs_blas1(INSData *data, Poisson_MF *poisson) {
  // Initialise cuBLAS
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
  // Make sure OP2 data is in the right place
  op_arg poisson_args[] = {
    op_arg_dat(poisson->uFluxX, -1, OP_ID, 21, "double", OP_READ),
    op_arg_dat(poisson->uFluxY, -1, OP_ID, 21, "double", OP_READ),
    op_arg_dat(poisson->qx, -1, OP_ID, 21, "double", OP_RW),
    op_arg_dat(poisson->qy, -1, OP_ID, 21, "double", OP_RW)
  };
  op_mpi_halo_exchanges_cuda(data->cells, 4, poisson_args);

  cublas_poisson_rhs1(handle, data->numCells, (double *)poisson->uFluxX->data_d,
                      (double *)poisson->uFluxY->data_d, (double *)poisson->qx->data_d,
                      (double *)poisson->qy->data_d);

  // Set correct dirty bits for OP2
  op_mpi_set_dirtybit_cuda(4, poisson_args);
  // Free resources used by cuBLAS
  hipblasDestroy(handle);
}
