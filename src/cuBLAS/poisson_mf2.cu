#include "hipblas.h"

#include "op_seq.h"
#include "../blas_calls.h"

// inline void cublas_poisson_rhs1(hipblasHandle_t handle, const int numCells,
//                                 const double *fluxX_d, const double *fluxY_d,
//                                 double *dudx_d, double *dudy_d, double *qx_d,
//                                 double *qy_d, double *gradx_d, double *grady_d) {
//   // HIPBLAS_OP_T because cublas is column major but constants are stored row major
//   double alpha = -1.0;
//   double alpha2 = 1.0;
//   double beta = 1.0;
//   double beta2 = 0.0;
//   hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 15, numCells, 15, &alpha2, constants->invMass_d, 15, dudx_d, 15, &beta2, gradx_d, 15);
//   hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 15, numCells, 15, &alpha2, constants->invMass_d, 15, dudy_d, 15, &beta2, grady_d, 15);
//
//   hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 15, numCells, 21, &alpha, constants->gInterp_d, 15, fluxX_d, 21, &beta, dudx_d, 15);
//   hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 15, numCells, 21, &alpha, constants->gInterp_d, 15, fluxY_d, 21, &beta, dudy_d, 15);
//
//   hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 15, numCells, 15, &alpha2, constants->invMass_d, 15, dudx_d, 15, &beta2, qx_d, 15);
//   hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 15, numCells, 15, &alpha2, constants->invMass_d, 15, dudy_d, 15, &beta2, qy_d, 15);
// }

void poisson_mf2_blas(INSData *data, Poisson_MF2 *poisson) {
  // Make sure OP2 data is in the right place
  op_arg poisson_args[] = {
    op_arg_dat(poisson->u, -1, OP_ID, 15, "double", OP_READ),
    op_arg_dat(poisson->op1, -1, OP_ID, 15 * 15, "double", OP_READ),
    op_arg_dat(poisson->rhs, -1, OP_ID, 15, "double", OP_WRITE)
  };
  op_mpi_halo_exchanges_cuda(data->cells, 3, poisson_args);

  // cublas_poisson_rhs1(constants->handle, data->numCells, (double *)poisson->u->data_d,
  //                     (double *)poisson->op1->data_d, (double *)poisson->rhs->data_d);

  // Set correct dirty bits for OP2
  op_mpi_set_dirtybit_cuda(3, poisson_args);
}
