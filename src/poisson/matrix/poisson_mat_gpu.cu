#include "poisson_mat.h"

#ifdef INS_MPI
#include "mpi_helper_func.h"
#endif

#include "dg_utils.h"

int PoissonMat::get_local_unknowns() {
  op_arg op2_args[] = {
    op_arg_dat(mesh->order, -1, OP_ID, 1, "int", OP_READ)
  };
  op_mpi_halo_exchanges_cuda(mesh->order->set, 1, op2_args);
  const int setSize = mesh->order->set->size;
  int *tempOrder = (int *)malloc(setSize * sizeof(int));
  hipMemcpy(tempOrder, mesh->order->data_d, setSize * sizeof(int), hipMemcpyDeviceToHost);
  int local_unkowns = 0;
  for(int i = 0; i < setSize; i++) {
    int Np, Nfp;
    DGUtils::basic_constants(tempOrder[i], &Np, &Nfp);
    local_unkowns += Np;
  }
  free(tempOrder);
  op_mpi_set_dirtybit_cuda(1, op2_args);
  return local_unkowns;
}

void PoissonSolve::set_glb_ind() {
  int global_ind = 0;
  #ifdef INS_MPI
  global_ind = get_global_mat_start_ind(unknowns);
  #endif
  op_arg args[] = {
    op_arg_dat(mesh->order, -1, OP_ID, 1, "int", OP_READ),
    op_arg_dat(glb_ind, -1, OP_ID, 1, "int", OP_WRITE)
  };
  op_mpi_halo_exchanges_cuda(mesh->cells, 2, args);

  const int setSize = mesh->cells->size;
  int *tempOrder = (int *)malloc(setSize * sizeof(int));
  hipMemcpy(tempOrder, mesh->order->data_d, setSize * sizeof(int), hipMemcpyDeviceToHost);
  int *data_ptr = (int *)malloc(setSize * sizeof(int));
  hipMemcpy(data_ptr, glb_ind->data_d, setSize * sizeof(int), hipMemcpyDeviceToHost);

  int ind = global_ind;
  for(int i = 0; i < mesh->cells->size; i++) {
    int Np, Nfp;
    DGUtils::basic_constants(tempOrder[i], &Np, &Nfp);
    data_ptr[i] = ind;
    ind += Np;
  }

  hipMemcpy(glb_ind->data_d, data_ptr, setSize * sizeof(int), hipMemcpyHostToDevice);

  op_mpi_set_dirtybit_cuda(2, args);
  free(data_ptr);
  free(tempOrder);
}