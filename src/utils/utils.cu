#include "op_seq.h"

#include <memory>

double *getOP2Array(op_dat dat) {
  op_arg args[] = {
    op_arg_dat(dat, -1, OP_ID, DG_SUB_CELLS, "double", OP_READ)
  };
  op_mpi_halo_exchanges_cuda(dat->set, 1, args);
  double *res = (double *)malloc(dat->set->size * DG_SUB_CELLS * sizeof(double));
  hipMemcpy(res, dat->data_d, dat->set->size * DG_SUB_CELLS * sizeof(double), hipMemcpyDeviceToHost);
  op_mpi_set_dirtybit_cuda(1, args);
  return res;
}
