#include "hypre_utils.h"

#include "op_seq.h"

#ifdef INS_MPI
#include "mpi_helper_func.h"

int get_global_start_ind(const int local_unknowns) {
  return get_global_mat_start_ind(local_unknowns);
}
#else
int get_global_start_ind(const int local_unknowns) {
  return 0;
}
#endif

// Get an OP2 dat as a HYPRE vector
void HYPREUtils::dat_to_new_vec(op_dat v_dat, HYPRE_IJVector *v,
                                const int local_unknowns) {
  int start_ind = get_global_start_ind(local_unknowns);
  int end_ind = start_ind + local_unknowns - 1;
  HYPRE_IJVectorCreate(MPI_COMM_WORLD, start_ind, end_ind, v);
  HYPRE_IJVectorSetObjectType(*v, HYPRE_PARCSR);
  HYPRE_IJVectorInitialize(*v);

  op_arg copy_args[] = {
    op_arg_dat(v_dat, -1, OP_ID, DG_NP, "double", OP_READ)
  };
  op_mpi_halo_exchanges_cuda(v_dat->set, 1, copy_args);

  int setSize = v_dat->set->size;

  double *v_data;
  hipMallocManaged(&v_data, DG_NP * setSize * sizeof(double));
  hipMemcpy(v_data, v_dat->data_d, DG_NP * setSize * sizeof(double), hipMemcpyDeviceToDevice);
  int *ind;
  hipMallocManaged(&ind, DG_NP * setSize * sizeof(int));
  for(int i = 0; i < DG_NP * setSize; i++) {
    ind[i] = start_ind + i;
  }

  HYPRE_IJVectorSetValues(*v, DG_NP * setSize, ind, v_data);

  hipFree(v_data);
  hipFree(ind);
  op_mpi_set_dirtybit_cuda(1, copy_args);

  HYPRE_IJVectorAssemble(*v);
}

// Get a HYPRE vector as an OP2 dat
void HYPREUtils::vec_to_dat(op_dat v_dat, HYPRE_IJVector *v,
                            const int local_unknowns) {
  op_arg copy_args[] = {
    op_arg_dat(v_dat, -1, OP_ID, DG_NP, "double", OP_WRITE)
  };
  op_mpi_halo_exchanges_cuda(v_dat->set, 1, copy_args);

  int setSize = v_dat->set->size;

  double *v_data;
  hipMallocManaged(&v_data, DG_NP * setSize * sizeof(double));
  int *ind;
  hipMallocManaged(&ind, DG_NP * setSize * sizeof(int));
  int start_ind = get_global_start_ind(local_unknowns);
  for(int i = 0; i < DG_NP * setSize; i++) {
    ind[i] = start_ind + i;
  }

  HYPRE_IJVectorGetValues(*v, DG_NP * setSize, ind, v_data);

  hipMemcpy(v_dat->data_d, v_data, DG_NP * setSize * sizeof(double), hipMemcpyDeviceToDevice);

  hipFree(v_data);
  hipFree(ind);
  op_mpi_set_dirtybit_cuda(1, copy_args);
}

void HYPREUtils::create_matrix(HYPRE_IJMatrix *mat, const int local_unknowns) {
  int start = 0;
  #ifdef INS_MPI
  start = get_global_mat_start_ind(local_unknowns);
  #endif
  HYPRE_IJMatrixCreate(MPI_COMM_WORLD, start, start + local_unknowns - 1, start, start + local_unknowns - 1, mat);
  HYPRE_IJMatrixSetObjectType(*mat, HYPRE_PARCSR);
  HYPRE_IJMatrixInitialize(*mat);
}
