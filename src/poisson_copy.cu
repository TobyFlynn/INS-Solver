#include "poisson.h"

// Copy u PETSc vec array to OP2 dat (TODO avoid this copy)
void Poisson_MF::copy_u(const double *u_d) {
  op_arg u_copy_args[] = {
    op_arg_dat(u, -1, OP_ID, 15, "double", OP_WRITE)
  };
  op_mpi_halo_exchanges_cuda(data->cells, 1, u_copy_args);
  hipMemcpy(u->data_d, u_d, data->numCells * 15 * sizeof(double), hipMemcpyDeviceToDevice);
  op_mpi_set_dirtybit_cuda(1, u_copy_args);
}

// Copy rhs OP2 dat to PETSc vec array (TODO avoid this copy)
void Poisson_MF::copy_rhs(double *rhs_d) {
  op_arg rhs_copy_args[] = {
    op_arg_dat(rhs, -1, OP_ID, 15, "double", OP_READ)
  };
  op_mpi_halo_exchanges_cuda(data->cells, 1, rhs_copy_args);
  hipMemcpy(rhs_d, rhs->data_d, data->numCells * 15 * sizeof(double), hipMemcpyDeviceToDevice);
  op_mpi_set_dirtybit_cuda(1, rhs_copy_args);
}

// Copy u PETSc vec array to OP2 dat (TODO avoid this copy)
void Poisson_MF2::copy_u(const double *u_d) {
  op_arg u_copy_args[] = {
    op_arg_dat(u, -1, OP_ID, 15, "double", OP_WRITE)
  };
  op_mpi_halo_exchanges_cuda(data->cells, 1, u_copy_args);
  hipMemcpy(u->data_d, u_d, data->numCells * 15 * sizeof(double), hipMemcpyDeviceToDevice);
  op_mpi_set_dirtybit_cuda(1, u_copy_args);
}

// Copy rhs OP2 dat to PETSc vec array (TODO avoid this copy)
void Poisson_MF2::copy_rhs(double *rhs_d) {
  op_arg rhs_copy_args[] = {
    op_arg_dat(rhs, -1, OP_ID, 15, "double", OP_READ)
  };
  op_mpi_halo_exchanges_cuda(data->cells, 1, rhs_copy_args);
  hipMemcpy(rhs_d, rhs->data_d, data->numCells * 15 * sizeof(double), hipMemcpyDeviceToDevice);
  op_mpi_set_dirtybit_cuda(1, rhs_copy_args);
}

// Create a PETSc vector for GPUs
void Poisson::create_vec(Vec *v, int size) {
  VecCreateSeqCUDA(PETSC_COMM_SELF, size * data->numCells, v);
}

// Destroy a PETSc vector
void Poisson::destroy_vec(Vec *v) {
  VecDestroy(v);
}

// Load a PETSc vector with values from an OP2 dat for GPUs
void Poisson::load_vec(Vec *v, op_dat v_dat, int size) {
  double *v_ptr;
  VecCUDAGetArray(*v, &v_ptr);
  op_arg vec_petsc_args[] = {
    op_arg_dat(v_dat, -1, OP_ID, size, "double", OP_READ)
  };
  op_mpi_halo_exchanges_cuda(data->cells, 1, vec_petsc_args);
  hipMemcpy(v_ptr, (double *)v_dat->data_d, size * data->numCells * sizeof(double), hipMemcpyDeviceToDevice);
  op_mpi_set_dirtybit_cuda(1, vec_petsc_args);
  VecCUDARestoreArray(*v, &v_ptr);
}

// Load an OP2 dat with the values from a PETSc vector for GPUs
void Poisson::store_vec(Vec *v, op_dat v_dat) {
  const double *v_ptr;
  VecCUDAGetArrayRead(*v, &v_ptr);
  op_arg vec_petsc_args[] = {
    op_arg_dat(v_dat, -1, OP_ID, 15, "double", OP_WRITE)
  };
  op_mpi_halo_exchanges_cuda(data->cells, 1, vec_petsc_args);
  hipMemcpy((double *)v_dat->data_d, v_ptr, 15 * data->numCells * sizeof(double), hipMemcpyDeviceToDevice);
  op_mpi_set_dirtybit_cuda(1, vec_petsc_args);
  VecCUDARestoreArrayRead(*v, &v_ptr);
}

// Create a PETSc matrix for GPUs
void Poisson::create_mat(Mat *m, int row, int col, int prealloc) {
  MatCreate(PETSC_COMM_SELF, m);
  MatSetSizes(*m, PETSC_DECIDE, PETSC_DECIDE, row, col);
  MatSetType(*m, MATSEQAIJCUSPARSE);
  MatSeqAIJSetPreallocation(*m, prealloc, NULL);
}

PetscErrorCode matAMult(Mat A, Vec x, Vec y) {
  timer->startLinearSolveMFMatMult();
  Poisson_MF *poisson;
  MatShellGetContext(A, &poisson);
  const double *x_ptr;
  double *y_ptr;
  VecCUDAGetArrayRead(x, &x_ptr);
  VecCUDAGetArray(y, &y_ptr);

  poisson->calc_rhs(x_ptr, y_ptr);

  VecCUDARestoreArrayRead(x, &x_ptr);
  VecCUDARestoreArray(y, &y_ptr);
  timer->endLinearSolveMFMatMult();
  return 0;
}

PetscErrorCode matAMult2(Mat A, Vec x, Vec y) {
  timer->startLinearSolveMFMatMult();
  Poisson_MF2 *poisson;
  MatShellGetContext(A, &poisson);
  const double *x_ptr;
  double *y_ptr;
  VecCUDAGetArrayRead(x, &x_ptr);
  VecCUDAGetArray(y, &y_ptr);

  poisson->calc_rhs(x_ptr, y_ptr);

  VecCUDARestoreArrayRead(x, &x_ptr);
  VecCUDARestoreArray(y, &y_ptr);
  timer->endLinearSolveMFMatMult();
  return 0;
}

void Poisson_MF::create_shell_mat(Mat *m) {
  MatCreateShell(PETSC_COMM_SELF, 15 * data->numCells, 15 * data->numCells, PETSC_DETERMINE, PETSC_DETERMINE, this, m);
  MatShellSetOperation(*m, MATOP_MULT, (void(*)(void))matAMult);
  MatShellSetVecType(*m, VECCUDA);
}

void Poisson_MF2::create_shell_mat(Mat *m) {
  MatCreateShell(PETSC_COMM_SELF, 15 * data->numCells, 15 * data->numCells, PETSC_DETERMINE, PETSC_DETERMINE, this, m);
  MatShellSetOperation(*m, MATOP_MULT, (void(*)(void))matAMult2);
  MatShellSetVecType(*m, VECCUDA);
}
