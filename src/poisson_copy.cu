#include "poisson.h"

#ifdef INS_MPI
#include "mpi_helper_func.h"
#endif

// Copy u PETSc vec array to OP2 dat (TODO avoid this copy)
void Poisson_MF2::copy_u(const double *u_d) {
  op_arg u_copy_args[] = {
    op_arg_dat(u, -1, OP_ID, 15, "double", OP_WRITE)
  };
  op_mpi_halo_exchanges_cuda(mesh->cells, 1, u_copy_args);
  hipMemcpy(u->data_d, u_d, u->set->size * 15 * sizeof(double), hipMemcpyDeviceToDevice);
  op_mpi_set_dirtybit_cuda(1, u_copy_args);
}

// Copy rhs OP2 dat to PETSc vec array (TODO avoid this copy)
void Poisson_MF2::copy_rhs(double *rhs_d) {
  op_arg rhs_copy_args[] = {
    op_arg_dat(rhs, -1, OP_ID, 15, "double", OP_READ)
  };
  op_mpi_halo_exchanges_cuda(mesh->cells, 1, rhs_copy_args);
  hipMemcpy(rhs_d, rhs->data_d, rhs->set->size * 15 * sizeof(double), hipMemcpyDeviceToDevice);
  op_mpi_set_dirtybit_cuda(1, rhs_copy_args);
}

// Create a PETSc vector for GPUs
void Poisson::create_vec(Vec *v, int size) {
  VecCreate(PETSC_COMM_WORLD, v);
  VecSetType(*v, VECCUDA);
  VecSetSizes(*v, size * mesh->cells->size, PETSC_DECIDE);
}

// Destroy a PETSc vector
void Poisson::destroy_vec(Vec *v) {
  VecDestroy(v);
}

// Load a PETSc vector with values from an OP2 dat for GPUs
void Poisson::load_vec(Vec *v, op_dat v_dat, int size) {
  double *v_ptr;
  VecCUDAGetArray(*v, &v_ptr);
  op_arg vec_petsc_args[] = {
    op_arg_dat(v_dat, -1, OP_ID, size, "double", OP_READ)
  };
  op_mpi_halo_exchanges_cuda(mesh->cells, 1, vec_petsc_args);
  hipMemcpy(v_ptr, (double *)v_dat->data_d, size * v_dat->set->size * sizeof(double), hipMemcpyDeviceToDevice);
  op_mpi_set_dirtybit_cuda(1, vec_petsc_args);
  VecCUDARestoreArray(*v, &v_ptr);
}

// Load an OP2 dat with the values from a PETSc vector for GPUs
void Poisson::store_vec(Vec *v, op_dat v_dat) {
  const double *v_ptr;
  VecCUDAGetArrayRead(*v, &v_ptr);
  op_arg vec_petsc_args[] = {
    op_arg_dat(v_dat, -1, OP_ID, 15, "double", OP_WRITE)
  };
  op_mpi_halo_exchanges_cuda(mesh->cells, 1, vec_petsc_args);
  hipMemcpy((double *)v_dat->data_d, v_ptr, 15 * v_dat->set->size * sizeof(double), hipMemcpyDeviceToDevice);
  op_mpi_set_dirtybit_cuda(1, vec_petsc_args);
  VecCUDARestoreArrayRead(*v, &v_ptr);
}

// Create a PETSc matrix for GPUs
void Poisson::create_mat(Mat *m, int row, int col, int prealloc0, int prealloc1) {
  MatCreate(PETSC_COMM_WORLD, m);
  MatSetSizes(*m, row, col, PETSC_DECIDE, PETSC_DECIDE);

  #ifdef INS_MPI
  MatSetType(*m, MATMPIAIJCUSPARSE);
  MatMPIAIJSetPreallocation(*m, prealloc0, NULL, prealloc1, NULL);
  #else
  MatSetType(*m, MATSEQAIJCUSPARSE);
  MatSeqAIJSetPreallocation(*m, prealloc0, NULL);
  #endif
  MatSetOption(*m, MAT_NEW_NONZERO_ALLOCATION_ERR, PETSC_FALSE);
}

PetscErrorCode matAMult2(Mat A, Vec x, Vec y) {
  timer->startLinearSolveMFMatMult();
  Poisson_MF2 *poisson;
  MatShellGetContext(A, &poisson);
  const double *x_ptr;
  double *y_ptr;
  VecCUDAGetArrayRead(x, &x_ptr);
  VecCUDAGetArray(y, &y_ptr);

  poisson->calc_rhs(x_ptr, y_ptr);

  VecCUDARestoreArrayRead(x, &x_ptr);
  VecCUDARestoreArray(y, &y_ptr);
  timer->endLinearSolveMFMatMult();
  return 0;
}

void Poisson_MF2::create_shell_mat(Mat *m) {
  MatCreateShell(PETSC_COMM_WORLD, 15 * mesh->cells->size, 15 * mesh->cells->size, PETSC_DETERMINE, PETSC_DETERMINE, this, m);
  MatShellSetOperation(*m, MATOP_MULT, (void(*)(void))matAMult2);
  MatShellSetVecType(*m, VECCUDA);
}
