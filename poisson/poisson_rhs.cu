#include "poisson_rhs.h"

#include "op_seq.h"

#include "operators.h"
#include "blas_calls.h"

#include "kernels/pRHS_faces.h"
#include "kernels/pRHS_bc.h"
#include "kernels/pRHS_du.h"
#include "kernels/pRHS_fluxq.h"
#include "kernels/pRHS_J.h"
#include "kernels/pRHS_qbc.h"

void poisson_rhs(const double *u, double *rhs) {
  op_arg u_copy_args[] = {
    op_arg_dat(data->pU, -1, OP_ID, 15, "double", OP_WRITE)
  };
  op_mpi_halo_exchanges_cuda(data->cells, 1, u_copy_args);
  hipMemcpy(data->pU->data_d, u, data->numCells * 15 * sizeof(double), hipMemcpyDeviceToDevice);
  op_mpi_set_dirtybit_cuda(1, u_copy_args);

  op_par_loop(pRHS_faces, "pRHS_faces", data->edges,
              op_arg_dat(data->edgeNum, -1, OP_ID, 2, "int", OP_READ),
              op_arg_dat(data->nodeX, -2, data->edge2cells, 3, "double", OP_READ),
              op_arg_dat(data->nodeY, -2, data->edge2cells, 3, "double", OP_READ),
              op_arg_dat(data->pU, -2, data->edge2cells, 15, "double", OP_READ),
              op_arg_dat(data->pExRHS[0], -2, data->edge2cells, 15, "double", OP_INC));

  op_par_loop(pRHS_bc, "pRHS_bc", data->bedges,
              op_arg_dat(data->bedge_type, -1, OP_ID, 1, "int", OP_READ),
              op_arg_dat(data->bedgeNum,   -1, OP_ID, 1, "int", OP_READ),
              op_arg_dat(data->pU, 0, data->bedge2cells, 15, "double", OP_READ),
              op_arg_dat(data->pExRHS[0], 0, data->bedge2cells, 15, "double", OP_INC));

  op_par_loop(pRHS_du, "pRHS_du", data->cells,
              op_arg_dat(data->nx, -1, OP_ID, 15, "double", OP_READ),
              op_arg_dat(data->ny, -1, OP_ID, 15, "double", OP_READ),
              op_arg_dat(data->fscale, -1, OP_ID, 15, "double", OP_READ),
              op_arg_dat(data->pU, -1, OP_ID, 15, "double", OP_READ),
              op_arg_dat(data->pExRHS[0], -1, OP_ID, 15, "double", OP_RW),
              op_arg_dat(data->pDu, -1, OP_ID, 15, "double", OP_WRITE),
              op_arg_dat(data->pFluxXu, -1, OP_ID, 15, "double", OP_WRITE),
              op_arg_dat(data->pFluxYu, -1, OP_ID, 15, "double", OP_WRITE));

  grad(data, data->pU, data->pDuDx, data->pDuDy);

  // qx and qy stored in pDuDx and pDuDy
  poisson_rhs_blas1(data);

  op_par_loop(pRHS_faces, "pRHS_faces", data->edges,
              op_arg_dat(data->edgeNum, -1, OP_ID, 2, "int", OP_READ),
              op_arg_dat(data->nodeX, -2, data->edge2cells, 3, "double", OP_READ),
              op_arg_dat(data->nodeY, -2, data->edge2cells, 3, "double", OP_READ),
              op_arg_dat(data->pDuDx, -2, data->edge2cells, 15, "double", OP_READ),
              op_arg_dat(data->pExRHS[0], -2, data->edge2cells, 15, "double", OP_INC));

  op_par_loop(pRHS_faces, "pRHS_faces", data->edges,
              op_arg_dat(data->edgeNum, -1, OP_ID, 2, "int", OP_READ),
              op_arg_dat(data->nodeX, -2, data->edge2cells, 3, "double", OP_READ),
              op_arg_dat(data->nodeY, -2, data->edge2cells, 3, "double", OP_READ),
              op_arg_dat(data->pDuDy, -2, data->edge2cells, 15, "double", OP_READ),
              op_arg_dat(data->pExRHS[1], -2, data->edge2cells, 15, "double", OP_INC));

  op_par_loop(pRHS_qbc, "pRHS_qbc", data->bedges,
              op_arg_dat(data->bedge_type, -1, OP_ID, 1, "int", OP_READ),
              op_arg_dat(data->bedgeNum,   -1, OP_ID, 1, "int", OP_READ),
              op_arg_dat(data->pDuDx, 0, data->bedge2cells, 15, "double", OP_READ),
              op_arg_dat(data->pExRHS[0], 0, data->bedge2cells, 15, "double", OP_INC));

  op_par_loop(pRHS_qbc, "pRHS_qbc", data->bedges,
              op_arg_dat(data->bedge_type, -1, OP_ID, 1, "int", OP_READ),
              op_arg_dat(data->bedgeNum,   -1, OP_ID, 1, "int", OP_READ),
              op_arg_dat(data->pDuDy, 0, data->bedge2cells, 15, "double", OP_READ),
              op_arg_dat(data->pExRHS[1], 0, data->bedge2cells, 15, "double", OP_INC));

  op_par_loop(pRHS_fluxq, "pRHS_fluxq", data->cells,
              op_arg_dat(data->nx, -1, OP_ID, 15, "double", OP_READ),
              op_arg_dat(data->ny, -1, OP_ID, 15, "double", OP_READ),
              op_arg_dat(data->fscale, -1, OP_ID, 15, "double", OP_READ),
              op_arg_dat(data->pTau, -1, OP_ID, 15, "double", OP_READ),
              op_arg_dat(data->pDu, -1, OP_ID, 15, "double", OP_READ),
              op_arg_dat(data->pDuDx, -1, OP_ID, 15, "double", OP_READ),
              op_arg_dat(data->pDuDy, -1, OP_ID, 15, "double", OP_READ),
              op_arg_dat(data->pExRHS[0], -1, OP_ID, 15, "double", OP_RW),
              op_arg_dat(data->pExRHS[1], -1, OP_ID, 15, "double", OP_RW),
              op_arg_dat(data->pFluxQ, -1, OP_ID, 15, "double", OP_WRITE));

  div(data, data->pDuDx, data->pDuDy, data->pDivQ);

  poisson_rhs_blas2(data);

  op_par_loop(pRHS_J, "pRHS_J", data->cells,
              op_arg_dat(data->J, -1, OP_ID, 15, "double", OP_READ),
              op_arg_dat(data->pRHSU, -1, OP_ID, 15, "double", OP_RW));

  op_arg rhs_copy_args[] = {
    op_arg_dat(data->pRHSU, -1, OP_ID, 15, "double", OP_READ)
  };
  op_mpi_halo_exchanges_cuda(data->cells, 1, rhs_copy_args);
  hipMemcpy(rhs, data->pRHSU->data_d, data->numCells * 15 * sizeof(double), hipMemcpyDeviceToDevice);
  op_mpi_set_dirtybit_cuda(1, rhs_copy_args);
}
