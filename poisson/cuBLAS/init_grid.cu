#include "hipblas.h"

#include "op_seq.h"
#include "../blas_calls.h"

inline void cublas_init_grid(hipblasHandle_t handle, const int numCells,
                        const double *node_coords, const int *cell2nodes,
                        double *x_d, double *y_d, double *xr_d, double *xs_d,
                        double *yr_d, double *ys_d) {
  double *ones_d;
  hipMalloc((void**)&ones_d, 15 * sizeof(double));
  hipMemcpy(ones_d, ones, 15 * sizeof(double), hipMemcpyHostToDevice);

  double *r_d;
  hipMalloc((void**)&r_d, 15 * sizeof(double));
  hipMemcpy(r_d, r, 15 * sizeof(double), hipMemcpyHostToDevice);

  double *s_d;
  hipMalloc((void**)&s_d, 15 * sizeof(double));
  hipMemcpy(s_d, s, 15 * sizeof(double), hipMemcpyHostToDevice);

  double *temp_d;
  hipMalloc((void**)&temp_d, numCells * 15 * sizeof(double));

  double *Dr_d;
  hipMalloc((void**)&Dr_d, 15 * 15 * sizeof(double));
  hipMemcpy(Dr_d, Dr, 15 * 15 * sizeof(double), hipMemcpyHostToDevice);

  double *Ds_d;
  hipMalloc((void**)&Ds_d, 15 * 15 * sizeof(double));
  hipMemcpy(Ds_d, Ds, 15 * 15 * sizeof(double), hipMemcpyHostToDevice);

  for(int c = 0; c < numCells; c++) {
    // Get nodes for this cell (on host)
    const double *n0 = &node_coords[2 * cell2nodes[3 * c]];
    const double *n1 = &node_coords[2 * cell2nodes[3 * c + 1]];
    const double *n2 = &node_coords[2 * cell2nodes[3 * c + 2]];

    double *temp = temp_d + c * 15;
    double *x = x_d + c * 15;
    double *y = y_d + c * 15;

    double alpha = 1.0;
    hipblasDcopy(handle, 15, ones_d, 1, x, 1);
    hipblasDaxpy(handle, 15, &alpha, r_d, 1, x, 1);
    alpha = 0.5 * n1[0];
    hipblasDscal(handle, 15, &alpha, x, 1);
    hipblasDcopy(handle, 15, ones_d, 1, temp, 1);
    alpha = 1.0;
    hipblasDaxpy(handle, 15, &alpha, s_d, 1, temp, 1);
    alpha = 0.5 * n2[0];
    hipblasDaxpy(handle, 15, &alpha, temp, 1, x, 1);
    hipblasDcopy(handle, 15, s_d, 1, temp, 1);
    alpha = 1.0;
    hipblasDaxpy(handle, 15, &alpha, r_d, 1, temp, 1);
    alpha = -0.5 * n0[0];
    hipblasDaxpy(handle, 15, &alpha, temp, 1, x, 1);

    hipblasDcopy(handle, 15, ones_d, 1, y, 1);
    alpha = 1.0;
    hipblasDaxpy(handle, 15, &alpha, r_d, 1, y, 1);
    alpha = 0.5 * n1[1];
    hipblasDscal(handle, 15, &alpha, y, 1);
    hipblasDcopy(handle, 15, ones_d, 1, temp, 1);
    alpha = 1.0;
    hipblasDaxpy(handle, 15, &alpha, s_d, 1, temp, 1);
    alpha = 0.5 * n2[1];
    hipblasDaxpy(handle, 15, &alpha, temp, 1, y, 1);
    hipblasDcopy(handle, 15, s_d, 1, temp, 1);
    alpha = 1.0;
    hipblasDaxpy(handle, 15, &alpha, r_d, 1, temp, 1);
    alpha = -0.5 * n0[1];
    hipblasDaxpy(handle, 15, &alpha, temp, 1, y, 1);
  }

  // HIPBLAS_OP_T because cublas is column major but constants are stored row major
  double alpha2 = 1.0;
  double beta = 0.0;
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 15, numCells, 15, &alpha2, Dr_d, 15, x_d, 15, &beta, xr_d, 15);
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 15, numCells, 15, &alpha2, Ds_d, 15, x_d, 15, &beta, xs_d, 15);
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 15, numCells, 15, &alpha2, Dr_d, 15, y_d, 15, &beta, yr_d, 15);
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 15, numCells, 15, &alpha2, Ds_d, 15, y_d, 15, &beta, ys_d, 15);

  hipFree(ones_d);
  hipFree(r_d);
  hipFree(temp_d);
  hipFree(Dr_d);
  hipFree(Ds_d);
}

void init_grid_blas(INSData *nsData) {
  // Initialise cuBLAS
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
  // Make sure OP2 data is in the right place
  op_arg init_grid_args[] = {
    op_arg_dat(nsData->x, -1, OP_ID, 15, "double", OP_WRITE),
    op_arg_dat(nsData->y, -1, OP_ID, 15, "double", OP_WRITE),
    op_arg_dat(nsData->xr, -1, OP_ID, 15, "double", OP_WRITE),
    op_arg_dat(nsData->xs, -1, OP_ID, 15, "double", OP_WRITE),
    op_arg_dat(nsData->yr, -1, OP_ID, 15, "double", OP_WRITE),
    op_arg_dat(nsData->ys, -1, OP_ID, 15, "double", OP_WRITE)
  };
  op_mpi_halo_exchanges_cuda(nsData->cells, 6, init_grid_args);

  cublas_init_grid(handle, nsData->numCells, (double *)nsData->node_coords->data,
                   (int *)nsData->cell2nodes->map, (double *)nsData->x->data_d,
                   (double *)nsData->y->data_d, (double *)nsData->xr->data_d,
                   (double *)nsData->xs->data_d, (double *)nsData->yr->data_d,
                   (double *)nsData->ys->data_d);

  // Set correct dirty bits for OP2
  op_mpi_set_dirtybit_cuda(6, init_grid_args);
  // Free resources used by cuBLAS
  hipblasDestroy(handle);
}
