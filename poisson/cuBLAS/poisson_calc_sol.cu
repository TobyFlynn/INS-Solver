#include "hipblas.h"

#include "op_seq.h"
#include "../blas_calls.h"

inline void cublas_poisson_calc_sol(hipblasHandle_t handle, const int numCells,
                                   double *sol_d) {
  double *invM_d;
  hipMalloc((void**)&invM_d, 15 * 15 * sizeof(double));
  hipMemcpy(invM_d, invM, 15 * 15 * sizeof(double), hipMemcpyHostToDevice);

  double *temp_d;
  hipMalloc((void**)&temp_d, 15 * numCells * sizeof(double));

  // HIPBLAS_OP_T because cublas is column major but constants are stored row major
  double alpha = 1.0;
  double beta = 0.0;
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 15, numCells, 15, &alpha, invM_d, 15, sol_d, 15, &beta, temp_d, 15);

  hipMemcpy(sol_d, temp_d, 15 * numCells * sizeof(double), hipMemcpyDeviceToDevice);

  hipFree(temp_d);
  hipFree(invM_d);
}

void poisson_calc_sol_blas(INSData *nsData) {
  // Initialise cuBLAS
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
  // Make sure OP2 data is in the right place
  op_arg poisson_sol_args[] = {
    op_arg_dat(nsData->sol, -1, OP_ID, 15, "double", OP_RW)
  };
  op_mpi_halo_exchanges_cuda(nsData->cells, 1, poisson_sol_args);

  cublas_poisson_calc_sol(handle, nsData->numCells, (double *)nsData->sol->data_d);

  // Set correct dirty bits for OP2
  op_mpi_set_dirtybit_cuda(1, poisson_sol_args);
  // Free resources used by cuBLAS
  hipblasDestroy(handle);
}
